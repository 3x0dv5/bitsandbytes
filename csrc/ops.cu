#include "hip/hip_runtime.h"
#include <ops.cuh>
#include <kernels.cuh>
#include <cub/device/device_scan.cuh>
#include <limits>
#include <cutlass/numeric_types.h>
#include <cutlass/gemm/device/gemm.h>



using std::cout;
using std::endl;

template <typename T> void estimateQuantiles(T *A, float *code, float offset, int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(code, 0, 256*sizeof(float)));
  kEstimateQuantiles<T><<<blocks, 512>>>(A, code, offset, std::numeric_limits<T>::max(), n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void quantize(float *code, float *A, unsigned char *out, int n)
{
  int blocks = n/1024;
  blocks = n % 1024 == 0 ? blocks : blocks + 1;
  kQuantize<<<blocks, 1024>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void dequantize(float *code, unsigned char *A, float *out, int n)
{
  int blocks = n/1024;
  blocks = n % 1024 == 0 ? blocks : blocks + 1;
  kDequantize<<<blocks, 1024>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <typename T, int STOCHASTIC> void quantizeBlockwise(float * code, T *A, float *absmax, unsigned char *out, float *rand, int rand_offset, const int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
  kQuantizeBlockwise<T, 4096, 4, STOCHASTIC><<<blocks, 1024>>>(code, A, absmax, out, rand, rand_offset, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template<typename T> void dequantizeBlockwise(float *code, unsigned char *A, float *absmax, T *out, int blocksize, const int n)
{
  int blocks = n/blocksize;
  blocks = n % blocksize == 0 ? blocks : blocks + 1;
  if(blocksize == 4096)
    kDequantizeBlockwise<T, 4096, 1024, 4><<<blocks, 4096/4>>>(code, A, absmax, out, n);
  else if(blocksize == 2048)
    kDequantizeBlockwise<T, 2048, 512, 4><<<blocks, 2048/4>>>(code, A, absmax, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template<typename T, int OPTIMIZER> void optimizer32bit(T* g, T* p, 
                float* state1, float* state2, float *unorm, float max_unorm, float param_norm,
                const float beta1, const float beta2, const float eps, const float weight_decay,
                const int step, const float lr, const float gnorm_scale, const int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
	switch(OPTIMIZER)
	{
		case ADAM:
      if(max_unorm > 0.0f)
			{ 
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
        kPreconditionOptimizer32bit2State<T, OPTIMIZER, 4096, 8><<<blocks, 512>>>(g, p, state1, state2, unorm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
      }
			kOptimizer32bit2State<T, OPTIMIZER><<<blocks, 1024>>>(g, p, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
		case MOMENTUM:
    case RMSPROP:
      if(max_unorm > 0.0f)
			{ 
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
				kPreconditionOptimizer32bit1State<T, OPTIMIZER, 4096, 8><<<blocks, 512>>>(g, p, state1, unorm, beta1, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
			}

			kOptimizer32bit1State<T, OPTIMIZER><<<blocks, 1024>>>(g, p, state1, unorm, max_unorm, param_norm, beta1, eps, weight_decay, step, lr, gnorm_scale, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
	}
}

template<typename T, int OPTIMIZER> void optimizerStatic8bit(T* p, T* g,
                unsigned char* state1, unsigned char* state2,
                float *unorm, float max_unorm, float param_norm,
                float beta1, float beta2,
                float eps, int step, float lr, 
                float* quantiles1, float* quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                float weight_decay,
                const float gnorm_scale, int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;

  if(max_unorm > 0.0f){ CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float))); }

	switch(OPTIMIZER)
	{
		case ADAM:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			CUDA_CHECK_RETURN(hipMemset(new_max2, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit2State<T, OPTIMIZER><<<blocks, 256>>>(p, g, state1, state2, unorm, beta1, beta2, eps, step, quantiles1, quantiles2, max1, max2, new_max1, new_max2, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit2State<T, OPTIMIZER><<<blocks, 1024>>>(p, g, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, eps, step, lr,
																														quantiles1, quantiles2, max1, max2, new_max1, new_max2, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
    case RMSPROP:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit1State<T, OPTIMIZER><<<blocks, 256>>>(p, g, state1, unorm, beta1, eps, step, quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit1State<T, OPTIMIZER><<<blocks, 1024>>>(p, g, state1, unorm, max_unorm, param_norm, beta1, eps, step, lr,
																														quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
		default:
			break;
	}
}

#define BLOCKSIZE_2STATE 2048
#define NUM_2STATE 8
#define BLOCKSIZE_1STATE 2048
#define NUM_1STATE 8

template<typename T, int OPTIMIZER> void optimizerStatic8bitBlockwise(T* p, T* g,
                unsigned char* state1, unsigned char* state2, float beta1, float beta2, float eps, int step, float lr, 
                float* quantiles1, float* quantiles2, float* absmax1, float* absmax2, float weight_decay, const float gnorm_scale, int n)
{

	int blocks = 0;
	switch(OPTIMIZER)
	{
		case ADAM:
			blocks = n/BLOCKSIZE_2STATE;
			blocks = n % BLOCKSIZE_2STATE == 0 ? blocks : blocks + 1;
			kOptimizerStatic8bit2StateBlockwise<T, OPTIMIZER, BLOCKSIZE_2STATE, NUM_2STATE><<<blocks, BLOCKSIZE_2STATE/NUM_2STATE>>>(p, g, state1, state2, beta1, beta2, eps, step, lr,
																														quantiles1, quantiles2, absmax1, absmax2, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
		case RMSPROP:
			blocks = n/BLOCKSIZE_1STATE;
			blocks = n % BLOCKSIZE_1STATE == 0 ? blocks : blocks + 1;
			kOptimizerStatic8bit1StateBlockwise<T, OPTIMIZER, BLOCKSIZE_1STATE, NUM_1STATE><<<blocks, BLOCKSIZE_1STATE/NUM_1STATE>>>(p, g, state1, beta1, beta2, eps, step, lr,
																														quantiles1, absmax1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
	}
}



template<typename T> void percentileClipping(T * g, float *gnorm_vec, int step, const int n)
{
  int blocks = n/2048;
  blocks = n % 2048 == 0 ? blocks : blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(&gnorm_vec[step % 100], 0, 1*sizeof(float)));
  kPercentileClipping<T, 2048, 4><<<blocks, 512>>>(g, gnorm_vec, step, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

#define UNSIGNED_CHAR 0

void gemmex(Context *context, bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc)
{
  const int falpha = 1;
  const int fbeta = 0;
  const void * alpha = &falpha;
  const void * beta = &fbeta;
	hipblasStatus_t status;

			status = hipblasGemmEx(context->m_handle,
					transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					m, n,	k,
					alpha, A, HIP_R_8I, lda, B, HIP_R_8I, ldb, beta,
					C, HIP_R_32I, ldc,
          HIP_R_32I, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      std::cout << "CUBLAS ERROR: Status " << status << std::endl;
    }

}

void strided_gemmex(Context *context, bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc, 
                    long long int strideA, long long int strideB, long long int strideC, int batchCount)
{
  const int falpha = 1;
  const int fbeta = 0;
  const void * alpha = &falpha;
  const void * beta = &fbeta;
	hipblasStatus_t status;

  //cout << transposeA << transposeB << endl;
  //printf("%i %i %i\n", m,n,k);
  //printf("%i %i %i\n", lda,ldb,ldc);
  //printf("%i %i %i\n", strideA, strideB, strideC);
  //printf("%i\n", batchCount);

			status = hipblasGemmStridedBatchedEx(context->m_handle,
					transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					m, n,	k,
					alpha, A, HIP_R_8I, lda, (long long int)strideA, B, HIP_R_8I, ldb, (long long int)strideB, beta,
					C, HIP_R_32I, ldc, (long long int)strideC, batchCount,
          HIP_R_32I, HIPBLAS_GEMM_DEFAULT);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      std::cout << "CUBLAS ERROR: Status " << status << std::endl;
    }

}

void cutlass_hgemm(bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc)
{

  // Define the GEMM operation
  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,                           // ElementA
    cutlass::layout::ColumnMajor,              // LayoutA
    cutlass::half_t,                           // ElementB
    cutlass::layout::ColumnMajor,              // LayoutB
    cutlass::half_t,                           // ElementOutput
    cutlass::layout::ColumnMajor,              // LayoutOutput
    float,                                     // ElementAccumulator
    cutlass::arch::OpClassWmmaTensorOp,            // tag indicating Tensor Cores
    cutlass::arch::Sm75,                        // tag indicating target GPU compute architecture
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    //cutlass::gemm::GemmShape<16, 16, 16>
    cutlass::gemm::GemmShape<8, 32, 16>
  >;

  Gemm gemm_op;
  cutlass::Status status;

  float alpha = 1.0f;
  float beta = 0.0f;

  cutlass::half_t const *ptrA = (cutlass::half_t*)A;
  cutlass::half_t const *ptrB = (cutlass::half_t*)B;
  cutlass::half_t const *ptrC = (cutlass::half_t*)C;

  cutlass::half_t       *ptrD = (cutlass::half_t*)C;
	int ldd = ldc;

  //
  // Launch GEMM on the device
  //
  status = gemm_op({
    {m, n, k},
    {ptrA, lda},            // TensorRef to A device tensor
    {ptrB, ldb},            // TensorRef to B device tensor
    {ptrC, ldc},            // TensorRef to C device tensor
    {ptrD, ldd},            // TensorRef to D device tensor - may be the same as C
    {alpha, beta}           // epilogue operation arguments
    });

  if (status != cutlass::Status::kSuccess)
	{
		printf("ERROR\n");
  }
} 


void cutlass_igemm(bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc)
{

  // Define the GEMM operation
  //using Gemm = cutlass::gemm::device::Gemm<
  //  int8_t,
  //  cutlass::layout::ColumnMajor,              // LayoutA
  //  int8_t,
  //  cutlass::layout::ColumnMajor,              // LayoutB
  //  int32_t,
  //  cutlass::layout::ColumnMajor,              // LayoutOutput
  //  int32_t,                                     // ElementAccumulator
  //  cutlass::arch::OpClassWmmaTensorOp,            // tag indicating Tensor Cores
  //  cutlass::arch::Sm75,                        // tag indicating target GPU compute architecture
  //  cutlass::gemm::GemmShape<64, 128, 64>,
  //  cutlass::gemm::GemmShape<32, 32, 64>,
  //  cutlass::gemm::GemmShape<16, 16, 16>
  //  //cutlass::gemm::GemmShape<32, 32, 16>
  //>;

  using Gemm = cutlass::gemm::device::Gemm<
    int8_t,
    cutlass::layout::RowMajor,              // LayoutA
    int8_t,
    cutlass::layout::ColumnMajor,              // LayoutB
    int32_t,
    cutlass::layout::ColumnMajor,              // LayoutOutput
    int32_t,                                     // ElementAccumulator
    cutlass::arch::OpClassTensorOp,            // tag indicating Tensor Cores
    cutlass::arch::Sm75                        // tag indicating target GPU compute architecture
    //cutlass::gemm::GemmShape<64, 128, 64>,
    //cutlass::gemm::GemmShape<32, 32, 64>,
    //cutlass::gemm::GemmShape<16, 16, 16>
    //cutlass::gemm::GemmShape<32, 32, 16>
  >;

  Gemm gemm_op;
  cutlass::Status status;

  float alpha = 1.0f;
  float beta = 0.0f;

  int8_t const *ptrA = (int8_t*)A;
  int8_t const *ptrB = (int8_t*)B;
  int32_t const *ptrC = (int32_t*)C;

  int32_t       *ptrD = (int32_t*)C;
	int ldd = ldc;

  //
  // Launch GEMM on the device
  //
  status = gemm_op({
    {m, n, k},
    {ptrA, lda},            // TensorRef to A device tensor
    {ptrB, ldb},            // TensorRef to B device tensor
    {ptrC, ldc},            // TensorRef to C device tensor
    {ptrD, ldd},            // TensorRef to D device tensor - may be the same as C
    {alpha, beta}           // epilogue operation arguments
    });

  if (status != cutlass::Status::kSuccess)
	{
		printf("ERROR\n");
  }
} 

//==============================================================
//                   TEMPLATE DEFINITIONS
//==============================================================

template void estimateQuantiles(half *A, float *code, float offset, int n);
template void estimateQuantiles(float *A, float *code, float offset, int n);

template void quantizeBlockwise<half, 0>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<float, 0>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<half, 1>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<float, 1>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void dequantizeBlockwise<half>(float *code, unsigned char *A, float *absmax, half *out, int blocksize, const int n);
template void dequantizeBlockwise<float>(float *code, unsigned char *A, float *absmax, float *out, int blocksize, const int n);

#define MAKE_optimizer32bit(name, gtype) \
template void optimizer32bit<gtype, name>(gtype* g, gtype* p, \
                float* state1, float* state2, float* unorm, float max_unorm, float param_norm, \
                const float beta1, const float beta2, const float eps, const float weight_decay, \
                const int step, const float lr, const float gnorm_scale, const int n);

MAKE_optimizer32bit(ADAM, half)
MAKE_optimizer32bit(ADAM, float)
MAKE_optimizer32bit(MOMENTUM, half)
MAKE_optimizer32bit(MOMENTUM, float)
MAKE_optimizer32bit(RMSPROP, half)
MAKE_optimizer32bit(RMSPROP, float)

#define MAKE_optimizerStatic8bit(name, gtype) \
template void optimizerStatic8bit<gtype, name>(gtype* p, gtype* g, unsigned char* state1, unsigned char* state2, \
                float *unorm, float max_unorm, float param_norm, \
                float beta1, float beta2, \
                float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                float weight_decay, \
                const float gnorm_scale, int n); \

MAKE_optimizerStatic8bit(ADAM, half)
MAKE_optimizerStatic8bit(ADAM, float)
MAKE_optimizerStatic8bit(MOMENTUM, half)
MAKE_optimizerStatic8bit(MOMENTUM, float)
MAKE_optimizerStatic8bit(RMSPROP, half)
MAKE_optimizerStatic8bit(RMSPROP, float)

#define MAKE_optimizerStatic8bitBlockwise(gtype, optim_name) \
template void optimizerStatic8bitBlockwise<gtype, optim_name>(gtype* p, gtype* g, \
                unsigned char* state1, unsigned char* state2, float beta1, float beta2, float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, float* absmax1, float* absmax2, float weight_decay, const float gnorm_scale, int n); \

MAKE_optimizerStatic8bitBlockwise(half, ADAM);
MAKE_optimizerStatic8bitBlockwise(float, ADAM);
MAKE_optimizerStatic8bitBlockwise(half, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(float, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(half, RMSPROP);
MAKE_optimizerStatic8bitBlockwise(float, RMSPROP);

template void percentileClipping(float * g, float *gnorm_vec, int step, const int n);
template void percentileClipping(half * g, float *gnorm_vec, int step, const int n);
