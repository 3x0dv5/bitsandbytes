#include "hip/hip_runtime.h"
#include <ops.cuh>
#include <kernels.cuh>
#include <cub/device/device_scan.cuh>
#include <limits>
#include <cutlass/numeric_types.h>
#include <cutlass/gemm/device/gemm.h>
#include <cassert>

using std::cout;
using std::endl;

template <typename T> void estimateQuantiles(T *A, float *code, float offset, int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(code, 0, 256*sizeof(float)));
  kEstimateQuantiles<T><<<blocks, 512>>>(A, code, offset, std::numeric_limits<T>::max(), n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void quantize(float *code, float *A, unsigned char *out, int n)
{
  int blocks = n/1024;
  blocks = n % 1024 == 0 ? blocks : blocks + 1;
  kQuantize<<<blocks, 1024>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void dequantize(float *code, unsigned char *A, float *out, int n)
{
  int blocks = n/1024;
  blocks = n % 1024 == 0 ? blocks : blocks + 1;
  kDequantize<<<blocks, 1024>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <typename T, int STOCHASTIC> void quantizeBlockwise(float * code, T *A, float *absmax, unsigned char *out, float *rand, int rand_offset, const int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
  kQuantizeBlockwise<T, 4096, 4, STOCHASTIC><<<blocks, 1024>>>(code, A, absmax, out, rand, rand_offset, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template<typename T> void dequantizeBlockwise(float *code, unsigned char *A, float *absmax, T *out, int blocksize, const int n)
{
  int blocks = n/blocksize;
  blocks = n % blocksize == 0 ? blocks : blocks + 1;
  if(blocksize == 4096)
    kDequantizeBlockwise<T, 4096, 1024, 4><<<blocks, 4096/4>>>(code, A, absmax, out, n);
  else if(blocksize == 2048)
    kDequantizeBlockwise<T, 2048, 512, 4><<<blocks, 2048/4>>>(code, A, absmax, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template<typename T, int OPTIMIZER> void optimizer32bit(T* g, T* p, 
                float* state1, float* state2, float *unorm, float max_unorm, float param_norm,
                const float beta1, const float beta2, const float eps, const float weight_decay,
                const int step, const float lr, const float gnorm_scale, const int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
	switch(OPTIMIZER)
	{
		case ADAM:
      if(max_unorm > 0.0f)
			{ 
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
        kPreconditionOptimizer32bit2State<T, OPTIMIZER, 4096, 8><<<blocks, 512>>>(g, p, state1, state2, unorm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
      }
			kOptimizer32bit2State<T, OPTIMIZER><<<blocks, 1024>>>(g, p, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
		case MOMENTUM:
    case RMSPROP:
      if(max_unorm > 0.0f)
			{ 
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
				kPreconditionOptimizer32bit1State<T, OPTIMIZER, 4096, 8><<<blocks, 512>>>(g, p, state1, unorm, beta1, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
			}

			kOptimizer32bit1State<T, OPTIMIZER><<<blocks, 1024>>>(g, p, state1, unorm, max_unorm, param_norm, beta1, eps, weight_decay, step, lr, gnorm_scale, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
	}
}

template<typename T, int OPTIMIZER> void optimizerStatic8bit(T* p, T* g,
                unsigned char* state1, unsigned char* state2,
                float *unorm, float max_unorm, float param_norm,
                float beta1, float beta2,
                float eps, int step, float lr, 
                float* quantiles1, float* quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                float weight_decay,
                const float gnorm_scale, int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;

  if(max_unorm > 0.0f){ CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float))); }

	switch(OPTIMIZER)
	{
		case ADAM:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			CUDA_CHECK_RETURN(hipMemset(new_max2, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit2State<T, OPTIMIZER><<<blocks, 256>>>(p, g, state1, state2, unorm, beta1, beta2, eps, step, quantiles1, quantiles2, max1, max2, new_max1, new_max2, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit2State<T, OPTIMIZER><<<blocks, 1024>>>(p, g, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, eps, step, lr,
																														quantiles1, quantiles2, max1, max2, new_max1, new_max2, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
    case RMSPROP:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit1State<T, OPTIMIZER><<<blocks, 256>>>(p, g, state1, unorm, beta1, eps, step, quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit1State<T, OPTIMIZER><<<blocks, 1024>>>(p, g, state1, unorm, max_unorm, param_norm, beta1, eps, step, lr,
																														quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
		default:
			break;
	}
}

#define BLOCKSIZE_2STATE 2048
#define NUM_2STATE 8
#define BLOCKSIZE_1STATE 2048
#define NUM_1STATE 8

template<typename T, int OPTIMIZER> void optimizerStatic8bitBlockwise(T* p, T* g,
                unsigned char* state1, unsigned char* state2, float beta1, float beta2, float eps, int step, float lr, 
                float* quantiles1, float* quantiles2, float* absmax1, float* absmax2, float weight_decay, const float gnorm_scale, int n)
{

	int blocks = 0;
	switch(OPTIMIZER)
	{
		case ADAM:
			blocks = n/BLOCKSIZE_2STATE;
			blocks = n % BLOCKSIZE_2STATE == 0 ? blocks : blocks + 1;
			kOptimizerStatic8bit2StateBlockwise<T, OPTIMIZER, BLOCKSIZE_2STATE, NUM_2STATE><<<blocks, BLOCKSIZE_2STATE/NUM_2STATE>>>(p, g, state1, state2, beta1, beta2, eps, step, lr,
																														quantiles1, quantiles2, absmax1, absmax2, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
		case RMSPROP:
			blocks = n/BLOCKSIZE_1STATE;
			blocks = n % BLOCKSIZE_1STATE == 0 ? blocks : blocks + 1;
			kOptimizerStatic8bit1StateBlockwise<T, OPTIMIZER, BLOCKSIZE_1STATE, NUM_1STATE><<<blocks, BLOCKSIZE_1STATE/NUM_1STATE>>>(p, g, state1, beta1, beta2, eps, step, lr,
																														quantiles1, absmax1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
	}
}



template<typename T> void percentileClipping(T * g, float *gnorm_vec, int step, const int n)
{
  int blocks = n/2048;
  blocks = n % 2048 == 0 ? blocks : blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(&gnorm_vec[step % 100], 0, 1*sizeof(float)));
  kPercentileClipping<T, 2048, 4><<<blocks, 512>>>(g, gnorm_vec, step, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

#define UNSIGNED_CHAR 0

void gemmex(Context *context, bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc)
{
  const int falpha = 1;
  const int fbeta = 0;
  const void * alpha = &falpha;
  const void * beta = &fbeta;
	hipblasStatus_t status;

			status = hipblasGemmEx(context->m_handle,
					transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					m, n,	k,
					alpha, A, HIP_R_8I, lda, B, HIP_R_8I, ldb, beta,
					C, HIP_R_32I, ldc,
          HIP_R_32I, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      std::cout << "CUBLAS ERROR: Status " << status << std::endl;
    }

}

void strided_gemmex(Context *context, bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc, 
                    long long int strideA, long long int strideB, long long int strideC, int batchCount)
{
  const int falpha = 1;
  const int fbeta = 0;
  const void * alpha = &falpha;
  const void * beta = &fbeta;
	hipblasStatus_t status;

  //cout << transposeA << transposeB << endl;
  //printf("%i %i %i\n", m,n,k);
  //printf("%i %i %i\n", lda,ldb,ldc);
  //printf("%i %i %i\n", strideA, strideB, strideC);
  //printf("%i\n", batchCount);

			status = hipblasGemmStridedBatchedEx(context->m_handle,
					transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
					m, n,	k,
					alpha, A, HIP_R_8I, lda, (long long int)strideA, B, HIP_R_8I, ldb, (long long int)strideB, beta,
					C, HIP_R_32I, ldc, (long long int)strideC, batchCount,
          HIP_R_32I, HIPBLAS_GEMM_DEFAULT);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
      std::cout << "CUBLAS ERROR: Status " << status << std::endl;
    }

}

int roundoff(int v, int d) {
    return (v + d - 1) / d * d;
}

//void igemmLt(Context *context, bool transposeA, bool transposeB, int m, int n, int k, const void *A, const void *B, void *C, int lda, int ldb, int ldc)
//{
//
//	hipblasLtHandle_t ltHandle = (hipblasLtHandle_t)context->m_handle;
//
//	hipblasLtMatmulDesc_t matmulDesc = NULL;
//	hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
//	hipblasOperation_t opTranspose = HIPBLAS_OP_T;
//	const void *D = C;
//
//  const int falpha = 1;
//  const int fbeta = 0;
//  const void * alpha = &falpha;
//  const void * beta = &fbeta;
//
//	hipblasLtMatrixTransformDesc_t transformDesc = NULL;
//	int8_t *Atransform = NULL, *Btransform = NULL;
//	int32_t *Ctransform                   = NULL;
//	hipblasLtMatrixLayout_t AtransformDesc = NULL, BtransformDesc = NULL, CtransformDesc = NULL;
//	float transformAlpha = 1.0f, transformBeta = 0.0f;
//	hipblasLtOrder_t order_COL = HIPBLASLT_ORDER_COL;
//	hipblasLtOrder_t order_COL32       = CUBLASLT_ORDER_COL32;
//	hipblasLtOrder_t order_COL4_4R2_8C = CUBLASLT_ORDER_COL4_4R2_8C;
//
//	//int ldatransform = 32 * m;
//	//int ldbtransform = 32 * roundoff(n, 8);
//	//int ldctransform = 32 * m;
//
//	//checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Atransform), sizeof(int8_t) * roundoff(k, 32) / 32 * ldatransform));
//	//checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Btransform), sizeof(int8_t) * roundoff(k, 32) / 32 * ldbtransform));
//	//checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Ctransform), sizeof(int32_t) * roundoff(n, 32) / 32 * ldctransform));
//
//	//checkCublasStatus(hipblasLtMatrixTransformDescCreate(&transformDesc, HIP_R_32F));
//
//	checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I));
//	if(transposeA)
//		checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));
//	if(transposeB)
//		checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opTranspose, sizeof(opTranspose)));
//
//	// ---------------------------------------------------------------------------------------------
//	// create descriptors for original matrices
//
//	checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, m, k, lda));
//	checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, k, n, ldb));
//	checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, m, n, ldc));
//
//	// ---------------------------------------------------------------------------------------------
//	// create descriptors for transformed matrices
//
//	cout << m << " " << n << " " << k << endl;
//	checkCublasStatus(hipblasLtMatrixLayoutCreate(&AtransformDesc, HIP_R_8I, m, k, lda));
//	checkCublasStatus(hipblasLtMatrixLayoutCreate(&BtransformDesc, HIP_R_8I, n, k, ldb));
//	checkCublasStatus(hipblasLtMatrixLayoutCreate(&CtransformDesc, HIP_R_32I, m, n, ldc));
//
//	checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(AtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL, sizeof(order_COL)));
//	checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(BtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL, sizeof(order_COL)));
//	checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(CtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL, sizeof(order_COL)));
//
//	// no need to transform C matrix as beta is assumed to be 0
// checkCublasStatus(hipblasLtMatmul(ltHandle,
//  															 matmulDesc,
//  															 alpha,
//  															 A,
//  															 AtransformDesc,
//  															 B,
//  															 BtransformDesc,
//  															 beta,
//  															 D,
//  															 CtransformDesc,
//  															 C,
//  															 CtransformDesc,
//  															 NULL,
//  															 NULL,
//  															 0,
//  															 0));
// 
//}

typedef enum Transform_t
{
	ROW = 0,
	COL = 1,
  COL32 = 2,
  COL_TURING = 3,
  COL_AMPERE = 4,
} Transform_t;

template<int ORDER> hipblasLtOrder_t get_order()
{
	switch(ORDER)
	{
		case ROW:
      return HIPBLASLT_ORDER_ROW;
			break;
    case COL:
      return HIPBLASLT_ORDER_COL;
      break;
    case COL32:
      return CUBLASLT_ORDER_COL32;
      break;
    case COL_TURING:
      return CUBLASLT_ORDER_COL4_4R2_8C;
      break;
    case COL_AMPERE:
      return CUBLASLT_ORDER_COL32_2R_4R4;
      break;
  }
}

template hipblasLtOrder_t get_order<ROW>();
template hipblasLtOrder_t get_order<COL>();
template hipblasLtOrder_t get_order<COL32>();
template hipblasLtOrder_t get_order<COL_TURING>();
template hipblasLtOrder_t get_order<COL_AMPERE>();


template<int ORDER> int get_leading_dim(int dim1, int dim2)
{
	switch(ORDER)
	{
		case ROW:
      return dim2;
			break;
    case COL:
      return dim1;
      break;
    case COL32:
      // 32*row tiles
      return dim1*32;
      break;
    case COL_TURING:
      // 32*8 tiles
      //int num_blocks = (dim1 + (8 - (dim1 % 8)))/8;
      //return 32*(dim1 + (8 - (dim1 % 8)));
      //return 8*(dim1  + (32 - (dim1 % 32)));
      //return 8*(dim2 + (32 - (dim2 % 32)));
      //return 256;
      return 32*roundoff(dim1, 8);
      //return 32*roundoff(dim2, 8);
      //return 8*roundoff(dim2, 32);
      //return 8*roundoff(dim1, 32);
      break;
    case COL_AMPERE:
      // 32*32 tiles
      return 32*roundoff(dim1, 32);
      break;
  }
}

template int get_leading_dim<ROW>(int dim1, int dim2);
template int get_leading_dim<COL>(int dim1, int dim2);
template int get_leading_dim<COL32>(int dim1, int dim2);

template <typename T, int SRC, int TARGET, bool transpose, int DTYPE> void transform(hipblasLtHandle_t ltHandle, T *A, T *out, int dim1, int dim2)
{

  hipblasLtOrder_t orderA = get_order<SRC>();
  hipblasLtOrder_t orderOut = get_order<TARGET>();
  int ldA = get_leading_dim<SRC>(dim1, dim2);
  int ldOut = get_leading_dim<TARGET>(dim1, dim2);
  
  hipblasLtMatrixLayout_t A_desc = NULL, out_desc = NULL;
  hipblasLtMatrixTransformDesc_t A2Out_desc = NULL;
  hipblasOperation_t opTranspose = HIPBLAS_OP_T;
  float transformAlpha = 1.0f, transformBeta = 0.0f;


  if(DTYPE == 8)
  {
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&A_desc, HIP_R_8I, dim1, dim2, ldA));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&out_desc, HIP_R_8I, dim1, dim2, ldOut));
  }
  else if(DTYPE == 32)
  {
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&A_desc, HIP_R_32I, dim1, dim2, ldA));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&out_desc, HIP_R_32I, dim1, dim2, ldOut));
  }
  else
  {
    printf("ERROR WRONG TYPE FOR TRANSFORM: %i\n", DTYPE);
  }

  checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(A_desc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &orderA, sizeof(orderA)));
  checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(out_desc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &orderOut, sizeof(orderOut)));

  checkCublasStatus(hipblasLtMatrixTransformDescCreate(&A2Out_desc, HIP_R_32F));

  if(transpose){ checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(A2Out_desc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose))); }

  checkCublasStatus(hipblasLtMatrixTransform(ltHandle, A2Out_desc, &transformAlpha, A, A_desc, &transformBeta, NULL, NULL, out, out_desc, 0));

  if (A_desc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(A_desc));
  if (out_desc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(out_desc));
  if (A2Out_desc) checkCublasStatus(hipblasLtMatrixTransformDescDestroy(A2Out_desc));
}

template void transform<int8_t, ROW, COL, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, ROW, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL32, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int32_t, ROW, COL32, false, 32>(hipblasLtHandle_t ltHandle, int32_t *A, int32_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL_TURING, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, ROW, COL_AMPERE, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int8_t, COL32, ROW, false, 8>(hipblasLtHandle_t ltHandle, int8_t *A, int8_t *out, int dim1, int dim2);
template void transform<int32_t, COL32, ROW, false, 32>(hipblasLtHandle_t ltHandle, int32_t *A, int32_t *out, int dim1, int dim2);

void LtIgemm(hipblasLtHandle_t ltHandle,
                   int m,
                   int n,
                   int k,
                   const int8_t *A,
                   const int8_t *B,
                   int32_t *C,
                   int lda,
                   int ldb,
                   int ldc) 
{
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    int32_t alpha = 1, beta = 0;
    hipblasOperation_t opT = HIPBLAS_OP_T;
    //hipblasOperation_t opN = HIPBLAS_OP_N;
    hipblasLtOrder_t col32 = CUBLASLT_ORDER_COL32;
    //hipblasLtOrder_t row = HIPBLASLT_ORDER_ROW;
    hipblasLtOrder_t col_turing = CUBLASLT_ORDER_COL4_4R2_8C;

    //hipblasLtMatmulPreference_t pref = NULL;
    //checkCublasStatus(hipblasLtMatmulPreferenceCreate(&pref)
    //checkCublasStatus(cublasLtMatmulPreferenceInit(pref)

    checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opT, sizeof(opT)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, m, k, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, n, k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, m, n, ldc));

    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col32, sizeof(col32)));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col32, sizeof(col32)));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &col_turing, sizeof(col_turing)));

    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     matmulDesc,
                                     &alpha,
                                     A,
                                     Adesc,
                                     B,
                                     Bdesc,
                                     &beta,
                                     C,
                                     Cdesc,
                                     C,
                                     Cdesc,
                                     NULL,
                                     NULL,
                                     0,
                                     0));

    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (matmulDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));
}

void cutlass_igemm(bool transposeA, bool transposeB, int m, int n, int k, void *A, void *B, void *C, int lda, int ldb, int ldc)
{

  // Define the GEMM operation
  using Gemm = cutlass::gemm::device::Gemm<
    int8_t,
    cutlass::layout::ColumnMajor,              // LayoutA
    int8_t,
    cutlass::layout::ColumnMajor,              // LayoutB
    int32_t,
    cutlass::layout::ColumnMajor,              // LayoutOutput
    int32_t,                                     // ElementAccumulator
    cutlass::arch::OpClassWmmaTensorOp,            // tag indicating Tensor Cores
    cutlass::arch::Sm75,                        // tag indicating target GPU compute architecture
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<32, 32, 64>,
    cutlass::gemm::GemmShape<16, 16, 16>
    //cutlass::gemm::GemmShape<32, 32, 16>
  >;

  //using Gemm = cutlass::gemm::device::Gemm<
  //  int8_t,
  //  cutlass::layout::RowMajor,              // LayoutA
  //  int8_t,
  //  cutlass::layout::ColumnMajor,              // LayoutB
  //  int32_t,
  //  cutlass::layout::ColumnMajor,              // LayoutOutput
  //  int32_t,                                     // ElementAccumulator
  //  cutlass::arch::OpClassTensorOp,            // tag indicating Tensor Cores
  //  cutlass::arch::Sm75                        // tag indicating target GPU compute architecture
  //  //cutlass::gemm::GemmShape<64, 128, 64>,
  //  //cutlass::gemm::GemmShape<32, 32, 64>,
  //  //cutlass::gemm::GemmShape<16, 16, 16>
  //  //cutlass::gemm::GemmShape<32, 32, 16>
  //>;

  Gemm gemm_op;
  cutlass::Status status;

  int alpha = 1;
  int beta = 0;

  int8_t const *ptrA = (int8_t*)A;
  int8_t const *ptrB = (int8_t*)B;
  int32_t const *ptrC = (int32_t*)C;

  int32_t       *ptrD = (int32_t*)C;
	int ldd = ldc;

  //
  // Launch GEMM on the device
  //
  status = gemm_op({
    {m, n, k},
    {ptrA, lda},            // TensorRef to A device tensor
    {ptrB, ldb},            // TensorRef to B device tensor
    {ptrC, ldc},            // TensorRef to C device tensor
    {ptrD, ldd},            // TensorRef to D device tensor - may be the same as C
    {alpha, beta}           // epilogue operation arguments
    });

  if (status != cutlass::Status::kSuccess)
	{
		printf("ERROR\n");
  }
} 

int fill_up_to_nearest_multiple(int value, int multiple)
{
  return value + (value % multiple == 0 ? 0 : (multiple - (value % multiple)));
}

void dequant_mm_int32_fp16(int *A, float *rowStats, float *colStats, half *out, float* newRowStats, float* newcolStats, int numRows, int numCols)
{
  int threads = 512;
  int tileCols = fill_up_to_nearest_multiple(numCols, 32);
  int n = numRows*tileCols;
  int subtile_rows = 128;
  int tilesize = 32*subtile_rows;
  int num_blocks = numRows/subtile_rows;
  num_blocks += (numRows % subtile_rows == 0) ? 0 : 1;
  num_blocks = num_blocks*(tileCols/32);
  assert(threads <= tilesize);

  //cout << num_blocks << " blocks" << endl;

  kdequant_mm_int32_fp16<4, 128, 512><<<num_blocks, threads>>>(A, rowStats, colStats, out, newRowStats, newcolStats, numRows, numCols, tileCols, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

#define STATS_THREADS 64
#define STATS_ITEMS 4
#define STATS_ROWS 16
void getColRowStats(half * A, float *rowStats, float *colStats, int rows, int cols)
{
  int tile_cols = STATS_THREADS*STATS_ITEMS;
  int tiledCols = fill_up_to_nearest_multiple(cols, tile_cols);
  int tiledRows = fill_up_to_nearest_multiple(rows, STATS_ROWS);
  int num_blocks = (tiledCols/tile_cols) * (tiledRows/STATS_ROWS);

  kgetColRowStats<half, STATS_THREADS, STATS_ITEMS, STATS_ROWS, STATS_THREADS*STATS_ITEMS><<<num_blocks, STATS_THREADS>>>(A, rowStats, colStats, rows, cols, tiledRows, tiledCols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());

}

void doubleRowColQuant(half * A, float *rowStats, float *colStats, char *out_col_normed, char *out_row_normed, int rows, int cols)
{
  int threads = 64;
  int items_per_thread = 4;
  int tile_cols = threads*items_per_thread;
  int tile_rows = 16;
  int tiledCols = fill_up_to_nearest_multiple(cols, tile_cols);
  int tiledRows = fill_up_to_nearest_multiple(rows, tile_rows);
  int num_blocks = (tiledCols/tile_cols) * (tiledRows/tile_rows);

  //cout << cols << " " << tiledCols << " " << tiledRows << endl;
  //cout << "num blocks " << num_blocks << endl;

  //cout << A << " " << out_col_normed << endl;
  kDoubleRowColQuant<64, 4, 16, 64*4><<<num_blocks, threads>>>(A, rowStats, colStats, out_col_normed, out_row_normed, rows, cols, tiledCols);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

//==============================================================
//                   TEMPLATE DEFINITIONS
//==============================================================

template void estimateQuantiles(half *A, float *code, float offset, int n);
template void estimateQuantiles(float *A, float *code, float offset, int n);

template void quantizeBlockwise<half, 0>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<float, 0>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<half, 1>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<float, 1>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void dequantizeBlockwise<half>(float *code, unsigned char *A, float *absmax, half *out, int blocksize, const int n);
template void dequantizeBlockwise<float>(float *code, unsigned char *A, float *absmax, float *out, int blocksize, const int n);

#define MAKE_optimizer32bit(name, gtype) \
template void optimizer32bit<gtype, name>(gtype* g, gtype* p, \
                float* state1, float* state2, float* unorm, float max_unorm, float param_norm, \
                const float beta1, const float beta2, const float eps, const float weight_decay, \
                const int step, const float lr, const float gnorm_scale, const int n);

MAKE_optimizer32bit(ADAM, half)
MAKE_optimizer32bit(ADAM, float)
MAKE_optimizer32bit(MOMENTUM, half)
MAKE_optimizer32bit(MOMENTUM, float)
MAKE_optimizer32bit(RMSPROP, half)
MAKE_optimizer32bit(RMSPROP, float)

#define MAKE_optimizerStatic8bit(name, gtype) \
template void optimizerStatic8bit<gtype, name>(gtype* p, gtype* g, unsigned char* state1, unsigned char* state2, \
                float *unorm, float max_unorm, float param_norm, \
                float beta1, float beta2, \
                float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                float weight_decay, \
                const float gnorm_scale, int n); \

MAKE_optimizerStatic8bit(ADAM, half)
MAKE_optimizerStatic8bit(ADAM, float)
MAKE_optimizerStatic8bit(MOMENTUM, half)
MAKE_optimizerStatic8bit(MOMENTUM, float)
MAKE_optimizerStatic8bit(RMSPROP, half)
MAKE_optimizerStatic8bit(RMSPROP, float)

#define MAKE_optimizerStatic8bitBlockwise(gtype, optim_name) \
template void optimizerStatic8bitBlockwise<gtype, optim_name>(gtype* p, gtype* g, \
                unsigned char* state1, unsigned char* state2, float beta1, float beta2, float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, float* absmax1, float* absmax2, float weight_decay, const float gnorm_scale, int n); \

MAKE_optimizerStatic8bitBlockwise(half, ADAM);
MAKE_optimizerStatic8bitBlockwise(float, ADAM);
MAKE_optimizerStatic8bitBlockwise(half, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(float, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(half, RMSPROP);
MAKE_optimizerStatic8bitBlockwise(float, RMSPROP);

template void percentileClipping(float * g, float *gnorm_vec, int step, const int n);
template void percentileClipping(half * g, float *gnorm_vec, int step, const int n);
