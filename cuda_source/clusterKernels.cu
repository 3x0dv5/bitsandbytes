#include "hip/hip_runtime.h"
#include <clusterKernels.cuh>
//#include <hipcub/hipcub.hpp>
#include <cub/block/block_radix_sort.cuh>
#include <cub/warp/warp_reduce.cuh>
#include <cub/block/block_load.cuh>
#include <cub/block/block_discontinuity.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_reduce.cuh>
#include <hipcub/hipcub.hpp>

#define HLF_MAX 65504

// source: https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda
__device__ float atomicMax(float* address, float val) {
  int* address_as_i = reinterpret_cast<int*>(address);
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = atomicCAS(
        reinterpret_cast<int*>(address), assumed,
        __float_as_int(fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

/**
 * @brief Quantizes x with the quantization map smem_code.
 *
 * @return returns quantized number.
 */
__device__ unsigned char quantize(float* smem_code, float x)
{
    unsigned char pivot = 127;
    unsigned char upper_pivot = 255;
    unsigned char lower_pivot = 0;

    // i>>=1 = {64, 32, 16, 8, 4, 2, 1}
    for(int i = 64; i > 0; i>>=1)
    {
        if(x > smem_code[pivot])
        {
            lower_pivot = pivot;
            pivot+=i;
        }
        else
        {
            upper_pivot = pivot;
            pivot-=i;
        }
    }

    if(pivot == upper_pivot)
    {
        if(fabsf(smem_code[pivot]-x) <= fabsf(smem_code[lower_pivot]-x))
            return pivot;
        else
            return lower_pivot;
    }
    else if(pivot == lower_pivot)
    {
        if(fabsf(smem_code[upper_pivot]-x) <= fabsf(smem_code[lower_pivot]-x))
            return upper_pivot;
        else
            return lower_pivot;
    }
    else
    {
        if(fabsf(smem_code[pivot]-x) <= fabsf(smem_code[upper_pivot]-x))
            if(fabsf(smem_code[pivot]-x) <= fabsf(smem_code[lower_pivot]-x))
                return pivot;
            else
                return lower_pivot;
        else
            if(fabsf(smem_code[upper_pivot]-x) <= fabsf(smem_code[lower_pivot]-x))
                return upper_pivot;
            else
                return lower_pivot;
    }
}

#define TH 1024
#define NUM 4
#define NUM_BLOCK 4096

template<typename T>
__launch_bounds__(TH, 1)
__global__ void kEstimateQuantiles(T *__restrict__ const A, float *code, const float offset, const T max_val, const int n)
{
  const int n_full = (NUM_BLOCK*(n/NUM_BLOCK)) + (n % NUM_BLOCK == 0 ? 0 : NUM_BLOCK);
  int valid_items = (blockIdx.x+1 == gridDim.x) ? n - (blockIdx.x*NUM_BLOCK) : NUM_BLOCK;
  const int base_idx = (blockIdx.x * NUM_BLOCK);
  const float reciprocal_num_blocks = 1.0f/(n < 4096 ? 1.0f : (n/NUM_BLOCK));

  T vals[NUM];

  typedef hipcub::BlockRadixSort<T, TH, NUM, hipcub::NullType, 4, true, hipcub::BLOCK_SCAN_RAKING> BlockRadixSort;
  typedef hipcub::BlockLoad<T, TH, NUM, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;

  __shared__ union {
      typename LoadFloat::TempStorage loadf;
      typename BlockRadixSort::TempStorage sort;
      int smem_qidx[NUM_BLOCK];
  } temp_storage;

  if(threadIdx.x < 256 && blockIdx.x == 0)
    code[threadIdx.x] = 0.0f;

  __syncthreads();

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_BLOCK)
  {
      valid_items = n - i > NUM_BLOCK ? NUM_BLOCK : n - i;

      // do not process half-blocks
      if(valid_items < NUM_BLOCK && n > NUM_BLOCK){ continue; }

      #pragma unroll 4
      for(int j = 0; j < NUM; j++)
          vals[j] = max_val;

      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(A[i]), vals, valid_items);

      #pragma unroll 4
      for(int j = 0; j < NUM; j++)
          vals[j] *= reciprocal_num_blocks;


      __syncthreads();
      // sort into striped pattern to mitigate bank conflicts
      // striped pattern index for thread 0 [0, 1024, 2048, 3096]
      // striped pattern index for thread 1 [1, 1025, 2049, 3097]
      BlockRadixSort(temp_storage.sort).SortBlockedToStriped(vals);

      __syncthreads();
      for(int j = threadIdx.x; j < NUM_BLOCK; j+=blockDim.x)
          temp_storage.smem_qidx[j] = -1;

      if(threadIdx.x < 256)
      {
          float q_interval = (1.0f-(2.0f*offset))/255.0f;
          int local_idx = round(((offset+(threadIdx.x*q_interval))*(valid_items-1)));
          temp_storage.smem_qidx[local_idx] = threadIdx.x;
      }

      __syncthreads();

      for(int i = threadIdx.x; i < NUM_BLOCK; i+=blockDim.x)
      {
          if(temp_storage.smem_qidx[i] != -1)
              atomicAdd(&code[temp_storage.smem_qidx[i]], vals[i/TH]);
      }
  }
}


__launch_bounds__(TH, 4)
__global__ void kQuantize(float * code, float * __restrict__ const A, unsigned char *out, const int n)
{
  const int n_full = (NUM_BLOCK*(n/NUM_BLOCK)) + (n % NUM_BLOCK == 0 ? 0 : NUM_BLOCK);
  int valid_items = (blockIdx.x+1 == gridDim.x) ? n - (blockIdx.x*NUM_BLOCK) : NUM_BLOCK;
  const int base_idx = (blockIdx.x * NUM_BLOCK);

  float vals[NUM];
  unsigned char qvals[NUM];

  typedef hipcub::BlockLoad<float, TH, NUM, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockStore<unsigned char, TH, NUM, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;

  __shared__ typename LoadFloat::TempStorage loadf;
  __shared__ typename StoreChar::TempStorage storec;
  __shared__ float smem_code[256];

  if(threadIdx.x < 256)
    smem_code[threadIdx.x] = code[threadIdx.x];

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_BLOCK)
  {
      valid_items = n - i > NUM_BLOCK ? NUM_BLOCK : n - i;

      __syncthreads();
      LoadFloat(loadf).Load(&(A[i]), vals, valid_items);

     #pragma unroll 4
     for(int j = 0; j < NUM; j++)
        qvals[j] = quantize(smem_code, vals[j]);

      __syncthreads();
      StoreChar(storec).Store(&(out[i]), qvals, valid_items);
  }
}

__global__ void kDequantize(float *code, unsigned char *A, float *out, const int n)
{
	const unsigned int numThreads = blockDim.x * gridDim.x;
	const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	__shared__ float smem_code[256];
	if(threadIdx.x < 256)
	{
		smem_code[threadIdx.x] = code[threadIdx.x];
	}

	__syncthreads();

	for (int i = idx;i < n; i += numThreads)
	{
		out[i] = smem_code[A[i]];
	}
}



#define NUM_PER_THREAD 4

template<typename T, int OPTIMIZER>
__launch_bounds__(TH, 1)
__global__ void kOptimizer32bit2State(T* g, T* p, 
                float* state1, float* state2,
                const float beta1, const float beta2, const float eps, const float weight_decay,
                const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n)
{

  const int n_full = ((TH*NUM_PER_THREAD)*(n/(TH*NUM_PER_THREAD))) + (n % (TH*NUM_PER_THREAD) == 0 ? 0 : (TH*NUM_PER_THREAD));
  const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
  int valid_items = 0;

  T g_vals[NUM_PER_THREAD];
  T p_vals[NUM_PER_THREAD];

  float s1_vals[NUM_PER_THREAD];
  float s2_vals[NUM_PER_THREAD];

  const float correction1 = 1.0f - powf(beta1, step);
  const float correction2 = sqrtf(1.0f - powf(beta2, step));
  const float step_size = -lr*correction2/correction1;

  typedef hipcub::BlockLoad<T, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> Load;
  typedef hipcub::BlockStore<T, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> Store;

  typedef hipcub::BlockLoad<float, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockStore<float, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreFloat;

  __shared__ union {
      typename Load::TempStorage load;
      typename Store::TempStorage store;
      typename LoadFloat::TempStorage loadf;
      typename StoreFloat::TempStorage storef;
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*TH*NUM_PER_THREAD)
  {
      valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

      __syncthreads();
      Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state2[i]), s2_vals, valid_items);
      __syncthreads();
      Load(temp_storage.load).Load(&(p[i]), p_vals, valid_items);

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
        g_vals[j] = gnorm_scale*((float)g_vals[j]);

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
      {
          switch(OPTIMIZER)
          {
              case ADAM: 
                  if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                  {
                    s1_vals[j] = s1_vals[j]*beta1 + ((1.0f -beta1)*((float)g_vals[j]));
                    s2_vals[j] = s2_vals[j]*beta2 + ((1.0f -beta2)*(((float)g_vals[j])*((float)g_vals[j])));
                    p_vals[j] = ((float)p_vals[j]) + (step_size*(s1_vals[j]/(sqrtf(s2_vals[j])+(eps*correction2))));
                  }
                  break;
          }
      }

      __syncthreads();
      Store(temp_storage.store).Store(&(p[i]), p_vals, valid_items);
      __syncthreads();
      StoreFloat(temp_storage.storef).Store(&(state1[i]), s1_vals, valid_items);
      __syncthreads();
      StoreFloat(temp_storage.storef).Store(&(state2[i]), s2_vals, valid_items);
  }
}

template<typename T, int OPTIMIZER>
__launch_bounds__(TH, 1)
__global__ void kOptimizer32bit1State(T* g, T* p, 
                float* state1, 
                const float beta1, const float eps, const float weight_decay,
                const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n)
{

  const int n_full = ((TH*NUM_PER_THREAD)*(n/(TH*NUM_PER_THREAD))) + (n % (TH*NUM_PER_THREAD) == 0 ? 0 : (TH*NUM_PER_THREAD));
  const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
  int valid_items = 0;

  T g_vals[NUM_PER_THREAD];
  T p_vals[NUM_PER_THREAD];

  float s1_vals[NUM_PER_THREAD];

  typedef hipcub::BlockLoad<T, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> Load;
  typedef hipcub::BlockStore<T, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> Store;

  typedef hipcub::BlockLoad<float, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockStore<float, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreFloat;

  __shared__ union {
      typename Load::TempStorage load;
      typename Store::TempStorage store;
      typename LoadFloat::TempStorage loadf;
      typename StoreFloat::TempStorage storef;
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*TH*NUM_PER_THREAD)
  {
      valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

      __syncthreads();
      Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items);
      __syncthreads();
      Load(temp_storage.load).Load(&(p[i]), p_vals, valid_items);

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
        g_vals[j] = gnorm_scale*((float)g_vals[j]);

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
      {
          switch(OPTIMIZER)
          {
              case MOMENTUM: 
                  if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                  {
                    if(step == 1)
                      s1_vals[j] = (float)g_vals[j];
                    else
                      s1_vals[j] = s1_vals[j]*beta1 + ((float)g_vals[j]);
                    p_vals[j] = ((float)p_vals[j]) + (-lr*(s1_vals[j]));
                  }
                  break;
          }
      }

      __syncthreads();
      Store(temp_storage.store).Store(&(p[i]), p_vals, valid_items);
      __syncthreads();
      StoreFloat(temp_storage.storef).Store(&(state1[i]), s1_vals, valid_items);
  }
}



#define NUM8BIT 16
#define NUM_THREADS 256
#define NUM_PER_BLOCK 4096

template<typename T, int OPTIMIZER>
__global__ void
__launch_bounds__(NUM_THREADS, 2)
kPreconditionOptimizerStatic8bit2State(T* p, T* __restrict__ const g, unsigned char*__restrict__  const state1, unsigned char* __restrict__ const state2,
                const float beta1, const float beta2,
                const float eps, const int step,
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                const float gnorm_scale, const int n)
{
    const int n_full = gridDim.x * NUM_PER_BLOCK;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
    int valid_items = n - (blockIdx.x*NUM_PER_BLOCK) > NUM_PER_BLOCK ? NUM_PER_BLOCK : n - (blockIdx.x*NUM_PER_BLOCK);
    float g_val = 0.0f;
    float local_max_s1 = -FLT_MAX;
    float local_max_s2 = -FLT_MAX;

    float s2_vals[NUM8BIT];
    float s1_vals[NUM8BIT];
    T g_vals[NUM8BIT];
    unsigned char m_c1[NUM8BIT];
    unsigned char r_c2[NUM8BIT];

    typedef hipcub::BlockRadixSort<float, NUM_THREADS, NUM8BIT, hipcub::NullType, 6, true, hipcub::BLOCK_SCAN_RAKING> BlockRadixSort;
    typedef hipcub::BlockLoad<T, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadUInt8;
    typedef hipcub::BlockReduce<float, NUM_THREADS> BlockReduce;


    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadUInt8::TempStorage loadc;
        typename BlockRadixSort::TempStorage sort;
        typename BlockReduce::TempStorage reduce;
    } temp_storage;

    __shared__ float smem_quantiles1[256];
    __shared__ float smem_quantiles2[256];

    if(threadIdx.x < 256)
    {
        smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];
        smem_quantiles2[threadIdx.x] = quantiles2[threadIdx.x];
        if(blockIdx.x == 0)
        {
            if(threadIdx.x == 0)
            {
                new_max1[0] = 0.0f;
                new_max2[0] = 0.0f;
            }
        }
    }

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += NUM_THREADS*gridDim.x*NUM8BIT)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadUInt8(temp_storage.loadc).Load(&(state1[i]), m_c1, valid_items, 128);
        __syncthreads();
        LoadUInt8(temp_storage.loadc).Load(&(state2[i]), r_c2, valid_items, 128);
        __syncthreads();

        #pragma unroll 16
        for(int j = 0; j < NUM8BIT; j++)
        {
            g_val = g_vals[j];
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[m_c1[j]]*max1[0]*beta1;
            s1_vals[j] += (1.0f-beta1)*g_val;
            local_max_s1 = fmaxf(local_max_s1, fabsf(s1_vals[j]));
        }

        #pragma unroll 16
        for(int j = 0; j < NUM8BIT; j++)
        {
            g_val = g_vals[j];
            g_val *= gnorm_scale;
            s2_vals[j] = smem_quantiles2[r_c2[j]]*max2[0]*beta2;
            s2_vals[j] += (1.0f-beta2)*g_val*g_val;
            local_max_s2 = fmaxf(local_max_s2, fabsf(s2_vals[j]));
        }
        __syncthreads();
    }

    local_max_s1 = BlockReduce(temp_storage.reduce).Reduce(local_max_s1, hipcub::Max(), valid_items);
    __syncthreads();
    local_max_s2 = BlockReduce(temp_storage.reduce).Reduce(local_max_s2, hipcub::Max(), valid_items);

    if(threadIdx.x == 0)
    {
        atomicMax(&new_max1[0], local_max_s1);
        atomicMax(&new_max2[0], local_max_s2);
    }
}

#define NUM_PER_THREAD2 4
#define NUM_THREADS2 1024
#define NUM_PER_BLOCK2 4096

template<typename T, int OPTIMIZER>
__global__ void
kOptimizerStatic8bit2State(T* p, T* const g, unsigned char* state1, unsigned char* state2,
                const float beta1, const float beta2,
                const float eps, const int step, const float lr,
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                float weight_decay,
                const float gnorm_scale, const int n)
{

    const int n_full = (blockDim.x * gridDim.x)*NUM_PER_THREAD2;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD2);
    int valid_items = 0;
    float g_val = 0.0f;
    float s1_vals[NUM_PER_THREAD2];
    float s2_vals[NUM_PER_THREAD2];
    const float correction1 = 1.0f - powf(beta1, step);
    const float correction2 = sqrtf(1.0f - powf(beta2, step));
    const float step_size = -lr*correction2/correction1;
    //const float step_size = -lr*correction2/correction1;
    float new_max_val1 = 1.0f/new_max1[0];
    float new_max_val2 = 1.0f/new_max2[0];

    unsigned char c1s[NUM_PER_THREAD2];
    unsigned char c2s[NUM_PER_THREAD2];
    T p_vals[NUM_PER_THREAD2];
    T g_vals[NUM_PER_THREAD2];
    typedef hipcub::BlockLoad<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;

    typedef hipcub::BlockStore<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;
    typedef hipcub::BlockStore<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

    __shared__ float smem_quantiles1[256];
    __shared__ float smem_quantiles2[256];

    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadChar::TempStorage loadc;
        typename StoreChar::TempStorage storec;
        typename StoreT::TempStorage storeh;
    } temp_storage;

    if(threadIdx.x < 512)
    {
        if(threadIdx.x < 256)
            smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];
        else
            smem_quantiles2[threadIdx.x-256] = quantiles2[threadIdx.x-256];
    }

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_THREADS2*NUM_PER_THREAD2)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state1[i]), c1s, valid_items, 128);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state2[i]), c2s, valid_items, 128);
        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(p[i]), p_vals, valid_items);

        if((i + (threadIdx.x*NUM_PER_THREAD2) + NUM_PER_THREAD2) > n){ continue; }

        # pragma unroll 4
        for(unsigned int j = 0; j < NUM_PER_THREAD2; j++)
        {
            g_val = float(g_vals[j]);
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[c1s[j]];
            s1_vals[j] = s1_vals[j]*max1[0];

            s1_vals[j] = (s1_vals[j]*beta1) + (((1.0f-beta1)*g_val));

            c1s[j] = quantize(smem_quantiles1, s1_vals[j]*new_max_val1);

            // make sure state1 term has still the same sign after quantization
            // (not needed for state2 term which has only positive values)
            if(signbit(smem_quantiles1[c1s[j]]) != signbit(s1_vals[j]))
            {
              if(s1_vals[j] > 0.0f)
                  c1s[j] += 1;
              else
                  c1s[j] -= 1;
            }

            s2_vals[j] = smem_quantiles2[c2s[j]];
            s2_vals[j] = s2_vals[j]*max2[0];
            s2_vals[j] = (s2_vals[j]*beta2) + (((1.0f-beta2)*g_val*g_val));
            c2s[j] = quantize(smem_quantiles2, s2_vals[j]*new_max_val2);
        }

        # pragma unroll 4
        for(unsigned int j = 0; j < NUM_PER_THREAD2; j++)
        {
            p_vals[j] = (T)(((float)p_vals[j]) + ((step_size*(s1_vals[j]/(sqrtf(s2_vals[j])+(correction2*eps))))));
            if(weight_decay > 0.0f)
                p_vals[j] = ((float)p_vals[j])*(1.0f-(lr*weight_decay));
        }

        StoreT(temp_storage.storeh).Store(&(p[i]), p_vals, valid_items);
        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state1[i]), c1s, valid_items);
        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state2[i]), c2s, valid_items);
        __syncthreads();
    }
}


template<typename T, int OPTIMIZER>
__global__ void
__launch_bounds__(NUM_THREADS, 2)
kPreconditionOptimizerStatic8bit1State(T* p, T* __restrict__ const g, unsigned char*__restrict__  const state1, 
                const float beta1, 
                const float eps, const int step,
                float* __restrict__ const quantiles1, 
                float* max1, float* new_max1, 
                const float gnorm_scale, const int n)
{
    const int n_full = gridDim.x * NUM_PER_BLOCK;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
    int valid_items = n - (blockIdx.x*NUM_PER_BLOCK) > NUM_PER_BLOCK ? NUM_PER_BLOCK : n - (blockIdx.x*NUM_PER_BLOCK);
    float g_val = 0.0f;
    float local_max_s1 = -FLT_MAX;

    float s1_vals[NUM8BIT];
    T g_vals[NUM8BIT];
    unsigned char m_c1[NUM8BIT];

    typedef hipcub::BlockRadixSort<float, NUM_THREADS, NUM8BIT, hipcub::NullType, 6, true, hipcub::BLOCK_SCAN_RAKING> BlockRadixSort;
    typedef hipcub::BlockLoad<T, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadUInt8;
    typedef hipcub::BlockReduce<float, NUM_THREADS> BlockReduce;


    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadUInt8::TempStorage loadc;
        typename BlockRadixSort::TempStorage sort;
        typename BlockReduce::TempStorage reduce;
    } temp_storage;

    __shared__ float smem_quantiles1[256];

    if(threadIdx.x < 256)
    {
        smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];
        if(blockIdx.x == 0)
            if(threadIdx.x == 0)
                new_max1[0] = 0.0f;
    }

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_THREADS*NUM8BIT)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadUInt8(temp_storage.loadc).Load(&(state1[i]), m_c1, valid_items, 128);

        #pragma unroll 16
        for(int j = 0; j < NUM8BIT; j++)
        {
            g_val = g_vals[j];
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[m_c1[j]]*max1[0];
            switch(OPTIMIZER)
            {
                case MOMENTUM: 
                    //TODO: if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                    //{
                      if(step == 1)
                        s1_vals[j] = (float)g_vals[j];
                      else
                        s1_vals[j] = s1_vals[j]*beta1 + ((float)g_vals[j]);
                    //}
                    break;
            }

            local_max_s1 = fmaxf(local_max_s1, fabsf(s1_vals[j]));
        }
    }

    __syncthreads();
    local_max_s1 = BlockReduce(temp_storage.reduce).Reduce(local_max_s1, hipcub::Max(), valid_items);
    if(threadIdx.x == 0){ atomicMax(&new_max1[0], local_max_s1); }

}

template<typename T, int OPTIMIZER>
__global__ void
kOptimizerStatic8bit1State(T* p, T* const g, unsigned char* state1,
                const float beta1, 
                const float eps, const int step, const float lr,
                float* __restrict__ const quantiles1, 
                float* max1, float* new_max1, 
                float weight_decay,
                const float gnorm_scale, const int n)
{

    const int n_full = (blockDim.x * gridDim.x)*NUM_PER_THREAD2;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD2);
    int valid_items = 0;
    float g_val = 0.0f;
    float s1_vals[NUM_PER_THREAD2];
    float new_max_val1 = 1.0f/new_max1[0];

    unsigned char c1s[NUM_PER_THREAD2];
    T p_vals[NUM_PER_THREAD2];
    T g_vals[NUM_PER_THREAD2];
    typedef hipcub::BlockLoad<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;

    typedef hipcub::BlockStore<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;
    typedef hipcub::BlockStore<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

    __shared__ float smem_quantiles1[256];

    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadChar::TempStorage loadc;
        typename StoreChar::TempStorage storec;
        typename StoreT::TempStorage storeh;
    } temp_storage;

    if(threadIdx.x < 256)
        smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_THREADS2*NUM_PER_THREAD2)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state1[i]), c1s, valid_items, 128);
        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(p[i]), p_vals, valid_items);

        if((i + (threadIdx.x*NUM_PER_THREAD2) + NUM_PER_THREAD2) > n){ continue; }

        # pragma unroll 4
        for(unsigned int j = 0; j < NUM_PER_THREAD2; j++)
        {
            g_val = float(g_vals[j]);
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[c1s[j]]*max1[0];

            switch(OPTIMIZER)
            {
                case MOMENTUM: 
                    //TODO: if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                    //{
                      if(step == 1)
                        s1_vals[j] = g_vals[j];
                      else
                        s1_vals[j] = s1_vals[j]*beta1 + ((float)g_vals[j]);

                      //TODO: if(weight_decay > 0.0f)
                      p_vals[j] = ((float)p_vals[j]) + (-lr*(s1_vals[j]));
                    //}
                    break;
            }

            c1s[j] = quantize(smem_quantiles1, s1_vals[j]*new_max_val1);

            // make sure state1 term has still the same sign after quantization
            if(signbit(smem_quantiles1[c1s[j]]) != signbit(s1_vals[j]))
            {
              if(s1_vals[j] > 0.0f)
                  c1s[j] += 1;
              else
                  c1s[j] -= 1;
            }
        }

        StoreT(temp_storage.storeh).Store(&(p[i]), p_vals, valid_items);
        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state1[i]), c1s, valid_items);
        __syncthreads();
    }
}


template<typename T, int BLOCK_SIZE, int NUM_VALS>
__global__ void kPercentileClipping(T * __restrict__ g, float *gnorm_vec, int step, const int n)
{
  const int n_full = (BLOCK_SIZE*(n/BLOCK_SIZE)) + (n % BLOCK_SIZE == 0 ? 0 : BLOCK_SIZE);
  int valid_items = 0;

  typedef hipcub::BlockReduce<float, BLOCK_SIZE/NUM_VALS> BlockReduce;
  typedef hipcub::BlockLoad<T, BLOCK_SIZE/NUM_VALS, NUM_VALS, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;

  __shared__ typename BlockReduce::TempStorage reduce;

  __shared__ typename LoadT::TempStorage loadT;
  T vals[NUM_VALS];
  float local_sum = 0.0f;

  if(blockIdx.x == 0 and threadIdx.x == 0)
    gnorm_vec[step % 100] = 0.0f;

  for (unsigned int i = (blockIdx.x * BLOCK_SIZE); i < n_full; i += gridDim.x*BLOCK_SIZE)
  {
      valid_items = n - i > BLOCK_SIZE ? BLOCK_SIZE : n - i;
      local_sum = 0.0f;

      __syncthreads();
      LoadT(loadT).Load(&(g[i]), vals, valid_items, (T)0.0f);

     #pragma unroll NUM_VALS
     for(int j = 0; j < NUM_VALS; j++)
       local_sum += ((float)vals[j])*((float)vals[j]);

    local_sum = BlockReduce(reduce).Sum(local_sum, valid_items);
    if(threadIdx.x == 0)
    {
      if(step == 1)
      {
        // initialize with the same norm for all positions
        //#pragma unroll 10
        for(int j = 0; j < 100; j++)
          atomicAdd(&gnorm_vec[j], local_sum);
      }
      else
          atomicAdd(&gnorm_vec[step % 100], local_sum);
    }

  }
}

//==============================================================
//                   TEMPLATE DEFINITIONS
//==============================================================


template __global__ void kEstimateQuantiles(float *__restrict__ const A, float *code, const float offset, const float max_val, const int n);
template __global__ void kEstimateQuantiles(half *__restrict__ const A, float *code, const float offset, const half max_val, const int n);

template __global__ void kOptimizer32bit1State<half, MOMENTUM>(half* g, half* p, float* state1, 
    const float beta1, const float eps, const float weight_decay,const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n);
template __global__ void kOptimizer32bit1State<float, MOMENTUM>(float* g, float* p, float* state1, 
    const float beta1, const float eps, const float weight_decay,const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n);

template __global__ void kOptimizer32bit2State<half, ADAM>(half* g, half* p, float* state1, float* state2,
    const float beta1, const float beta2, const float eps, const float weight_decay,const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n);
template __global__ void kOptimizer32bit2State<float, ADAM>(float* g, float* p, float* state1, float* state2,
    const float beta1, const float beta2, const float eps, const float weight_decay,const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n);

#define MAKE_PreconditionStatic8bit1State(oname, gtype) \
template __global__ void kPreconditionOptimizerStatic8bit1State<gtype, oname>(gtype* p, gtype* __restrict__ const g, unsigned char*__restrict__  const state1,  \
                const float beta1,  \
                const float eps, const int step,  \
                float* __restrict__ const quantiles1,  \
                float* max1, float* new_max1,  \
                const float gnorm_scale,  \
                const int n); \

MAKE_PreconditionStatic8bit1State(MOMENTUM, half)
MAKE_PreconditionStatic8bit1State(MOMENTUM, float)

#define MAKE_PreconditionStatic8bit2State(oname, gtype) \
template __global__ void kPreconditionOptimizerStatic8bit2State<gtype, oname>(gtype* p, gtype* __restrict__ const g, unsigned char*__restrict__  const state1, unsigned char* __restrict__ const state2, \
                const float beta1, const float beta2, \
                const float eps, const int step,  \
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                const float gnorm_scale,  \
                const int n); \

MAKE_PreconditionStatic8bit2State(ADAM, half)
MAKE_PreconditionStatic8bit2State(ADAM, float)


#define MAKE_optimizerStatic8bit1State(oname, gtype) \
template __global__ void kOptimizerStatic8bit1State<gtype, oname>(gtype* p, gtype* const g, unsigned char* state1,  \
                const float beta1,  \
                const float eps, const int step, const float lr, \
                float* __restrict__ const quantiles1,  \
                float* max1, float* new_max1,  \
                float weight_decay, \
                const float gnorm_scale,  \
                const int n); \

MAKE_optimizerStatic8bit1State(MOMENTUM, half)
MAKE_optimizerStatic8bit1State(MOMENTUM, float)

#define MAKE_optimizerStatic8bit2State(oname, gtype) \
template __global__ void kOptimizerStatic8bit2State<gtype, oname>(gtype* p, gtype* const g, unsigned char* state1, unsigned char* state2, \
                const float beta1, const float beta2, \
                const float eps, const int step, const float lr, \
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                float weight_decay, \
                const float gnorm_scale,  \
                const int n); \

MAKE_optimizerStatic8bit2State(ADAM, half)
MAKE_optimizerStatic8bit2State(ADAM, float)

template __global__ void kPercentileClipping<float, 2048, 4>(float * __restrict__ g, float *gnorm_vec, int step, const int n);
template __global__ void kPercentileClipping<half, 2048, 4>(half * __restrict__ g, float *gnorm_vec, int step, const int n);

