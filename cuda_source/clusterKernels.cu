#include "hip/hip_runtime.h"
#include <clusterKernels.cuh>
//#include <hipcub/hipcub.hpp>
#include <cub/block/block_radix_sort.cuh>
#include <cub/warp/warp_reduce.cuh>
#include <cub/block/block_load.cuh>
#include <cub/block/block_discontinuity.cuh>
#include <cub/block/block_store.cuh>

#define HLF_MAX 65504

template __global__ void kElementWise<ksmul>(const float *A, const float *B, float *out, const float scalar, int size);
template<int operation> __global__ void kElementWise(const float *A, const float *B, float *out, const float scalar, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
  {
	  //this switch operation will be removed by the compiler upon instantiation of the template
       switch(operation)
	   {
         case ksmul: out[i] = A[i] * scalar; break;
	   }
  }
}


/**
 * @brief Quantizes x with the quantization map smem_code.
 *
 * @return returns quantized number.
 */
__device__ unsigned char quantize(float* smem_code, float x)
{
    unsigned char pivot = 127;
    unsigned char upper_pivot = 255;
    unsigned char lower_pivot = 0;

    // i>>=1 = {64, 32, 16, 8, 4, 2, 1}
    for(int i = 64; i > 0; i>>=1)
    {
        if(x > smem_code[pivot])
        {
            lower_pivot = pivot;
            pivot+=i;
        }
        else
        {
            upper_pivot = pivot;
            pivot-=i;
        }
    }

    if(pivot == upper_pivot)
    {
        if(fabsf(smem_code[pivot]-x) <= fabsf(smem_code[lower_pivot]-x))
            return pivot;
        else
            return lower_pivot;
    }
    else if(pivot == lower_pivot)
    {
        if(fabsf(smem_code[upper_pivot]-x) <= fabsf(smem_code[lower_pivot]-x))
            return upper_pivot;
        else
            return lower_pivot;
    }
    else
    {
        if(fabsf(smem_code[pivot]-x) <= fabsf(smem_code[upper_pivot]-x))
            if(fabsf(smem_code[pivot]-x) <= fabsf(smem_code[lower_pivot]-x))
                return pivot;
            else
                return lower_pivot;
        else
            if(fabsf(smem_code[upper_pivot]-x) <= fabsf(smem_code[lower_pivot]-x))
                return upper_pivot;
            else
                return lower_pivot;
    }
}

#define TH 1024
#define NUM 4
#define NUM_BLOCK 4096

template __global__ void kEstimateQuantiles(float *__restrict__ const A, float *code, const float offset, const float max_val, const int n);
template __global__ void kEstimateQuantiles(half *__restrict__ const A, float *code, const float offset, const half max_val, const int n);
template<typename T>
__launch_bounds__(TH, 1)
__global__ void kEstimateQuantiles(T *__restrict__ const A, float *code, const float offset, const T max_val, const int n)
{
  const int n_full = (NUM_BLOCK*(n/NUM_BLOCK)) + (n % NUM_BLOCK == 0 ? 0 : NUM_BLOCK);
  int valid_items = (blockIdx.x+1 == gridDim.x) ? n - (blockIdx.x*NUM_BLOCK) : NUM_BLOCK;
  const int base_idx = (blockIdx.x * NUM_BLOCK);
  const float reciprocal_num_blocks = 1.0f/(n < 4096 ? 1.0f : (n/NUM_BLOCK));

  T vals[NUM];

  typedef hipcub::BlockRadixSort<T, TH, NUM, hipcub::NullType, 4, true, hipcub::BLOCK_SCAN_RAKING> BlockRadixSort;
  typedef hipcub::BlockLoad<T, TH, NUM, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;

  __shared__ union {
      typename LoadFloat::TempStorage loadf;
      typename BlockRadixSort::TempStorage sort;
      int smem_qidx[NUM_BLOCK];
  } temp_storage;

  if(threadIdx.x < 256 && blockIdx.x == 0)
    code[threadIdx.x] = 0.0f;

  __syncthreads();

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_BLOCK)
  {
      valid_items = n - i > NUM_BLOCK ? NUM_BLOCK : n - i;

      // do not process half-blocks
      if(valid_items < NUM_BLOCK && n > NUM_BLOCK){ continue; }

      #pragma unroll 4
      for(int j = 0; j < NUM; j++)
          vals[j] = max_val;

      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(A[i]), vals, valid_items);

      #pragma unroll 4
      for(int j = 0; j < NUM; j++)
          vals[j] *= reciprocal_num_blocks;


      __syncthreads();
      // sort into striped pattern to mitigate bank conflicts
      // striped pattern index for thread 0 [0, 1024, 2048, 3096]
      // striped pattern index for thread 1 [1, 1025, 2049, 3097]
      BlockRadixSort(temp_storage.sort).SortBlockedToStriped(vals);

      __syncthreads();
      for(int j = threadIdx.x; j < NUM_BLOCK; j+=blockDim.x)
          temp_storage.smem_qidx[j] = -1;

      if(threadIdx.x < 256)
      {
          float q_interval = (1.0f-(2.0f*offset))/255.0f;
          int local_idx = round(((offset+(threadIdx.x*q_interval))*(valid_items-1)));
          temp_storage.smem_qidx[local_idx] = threadIdx.x;
      }

      __syncthreads();

      for(int i = threadIdx.x; i < NUM_BLOCK; i+=blockDim.x)
      {
          if(temp_storage.smem_qidx[i] != -1)
              atomicAdd(&code[temp_storage.smem_qidx[i]], vals[i/TH]);
      }
  }
}


__launch_bounds__(TH, 4)
__global__ void kQuantize(float * code, float * __restrict__ const A, unsigned char *out, const int n)
{
  const int n_full = (NUM_BLOCK*(n/NUM_BLOCK)) + (n % NUM_BLOCK == 0 ? 0 : NUM_BLOCK);
  int valid_items = (blockIdx.x+1 == gridDim.x) ? n - (blockIdx.x*NUM_BLOCK) : NUM_BLOCK;
  const int base_idx = (blockIdx.x * NUM_BLOCK);

  float vals[NUM];
  unsigned char qvals[NUM];

  typedef hipcub::BlockLoad<float, TH, NUM, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockStore<unsigned char, TH, NUM, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;

  __shared__ typename LoadFloat::TempStorage loadf;
  __shared__ typename StoreChar::TempStorage storec;
  __shared__ float smem_code[256];

  if(threadIdx.x < 256)
    smem_code[threadIdx.x] = code[threadIdx.x];

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_BLOCK)
  {
      valid_items = n - i > NUM_BLOCK ? NUM_BLOCK : n - i;

      __syncthreads();
      LoadFloat(loadf).Load(&(A[i]), vals, valid_items);

     #pragma unroll 4
     for(int j = 0; j < NUM; j++)
        qvals[j] = quantize(smem_code, vals[j]);

      __syncthreads();
      StoreChar(storec).Store(&(out[i]), qvals, valid_items);
  }
}

__global__ void kDequantize(float *code, unsigned char *A, float *out, const int n)
{
	const unsigned int numThreads = blockDim.x * gridDim.x;
	const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	__shared__ float smem_code[256];
	if(threadIdx.x < 256)
	{
		smem_code[threadIdx.x] = code[threadIdx.x];
	}

	__syncthreads();

	for (int i = idx;i < n; i += numThreads)
	{
		out[i] = smem_code[A[i]];
	}
}



#define NUM_PER_THREAD 4

template __global__ void kOptimizer_32bit_2State<float, adam>(float* g, float* p, float* state1, float* state2,
    const float beta1, const float beta2, const float eps, const float weight_decay,const int step, const float lr, const int n);
template<typename T, int OPTIMIZER>
__launch_bounds__(TH, 1)
__global__ void kOptimizer_32bit_2State(T* g, T* p, 
                float* state1, float* state2,
                const float beta1, const float beta2, const float eps, const float weight_decay,
                const int step, const float lr, const int n)
{

  const int n_full = ((TH*NUM_PER_THREAD)*(n/(TH*NUM_PER_THREAD))) + (n % (TH*NUM_PER_THREAD) == 0 ? 0 : (TH*NUM_PER_THREAD));
  const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
  int valid_items = 0;

  T g_vals[NUM_PER_THREAD];
  T p_vals[NUM_PER_THREAD];

  float s1_vals[NUM_PER_THREAD];
  float s2_vals[NUM_PER_THREAD];

  const float correction1 = 1.0f - powf(beta1, step);
  const float correction2 = sqrtf(1.0f - powf(beta2, step));
  const float step_size = -lr*correction2/correction1;

  typedef hipcub::BlockLoad<T, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> Load;
  typedef hipcub::BlockStore<T, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> Store;

  typedef hipcub::BlockLoad<float, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockStore<float, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreFloat;

  __shared__ union {
      typename Load::TempStorage load;
      typename Store::TempStorage store;
      typename LoadFloat::TempStorage loadf;
      typename StoreFloat::TempStorage storef;
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*TH*NUM_PER_THREAD)
  {
      valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : i + (TH*NUM_PER_THREAD) - n;

      __syncthreads();
      Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state2[i]), s2_vals, valid_items);
      __syncthreads();
      Load(temp_storage.load).Load(&(p[i]), p_vals, valid_items);

      if((i + (threadIdx.x*NUM_PER_THREAD) + NUM_PER_THREAD) <= n)
      {
        # pragma unroll 4
        for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
        {
            switch(OPTIMIZER)
            {
                case adam: 
                    s1_vals[j] = s1_vals[j]*beta1 + ((1.0f -beta1)*((float)g_vals[j]));
                    s2_vals[j] = s2_vals[j]*beta2 + ((1.0f -beta2)*(((float)g_vals[j])*((float)g_vals[j])));
                    p_vals[j] = ((float)p_vals[j]) + (step_size*(s1_vals[j]/(sqrtf(s2_vals[j])+(eps*correction2))));
                    break;
                case momentum: 
                    break;
            }
        }
      }

      __syncthreads();
      Store(temp_storage.store).Store(&(p[i]), p_vals, valid_items);
      __syncthreads();
      StoreFloat(temp_storage.storef).Store(&(state1[i]), s1_vals, valid_items);
      __syncthreads();
      StoreFloat(temp_storage.storef).Store(&(state2[i]), s2_vals, valid_items);
        __syncthreads();
  }
}
