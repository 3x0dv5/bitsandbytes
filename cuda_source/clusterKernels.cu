#include "hip/hip_runtime.h"
#include <clusterKernels.cuh>
//#include <hipcub/hipcub.hpp>
#include <cub/block/block_radix_sort.cuh>
#include <cub/warp/warp_reduce.cuh>
#include <cub/block/block_load.cuh>
#include <cub/block/block_discontinuity.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_reduce.cuh>
#include <hipcub/hipcub.hpp>

#define HLF_MAX 65504

// source: https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda
__device__ float atomicMax(float* address, float val) {
  int* address_as_i = reinterpret_cast<int*>(address);
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = atomicCAS(
        reinterpret_cast<int*>(address), assumed,
        __float_as_int(fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

__device__ float atomicMin(float* address, float val) {
  int* address_as_i = reinterpret_cast<int*>(address);
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = atomicCAS(
        reinterpret_cast<int*>(address), assumed,
        __float_as_int(fminf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

__device__ unsigned char quantize(float* smem_code, float x)
{
    int pivot = 127;
    int upper_pivot = 255;
    int lower_pivot = 0;

    float val = smem_code[pivot];
    float lower = -1.0f;
    float upper = 1.0f;

    // i>>=1 = {64, 32, 16, 8, 4, 2, 1}
    for(int i = 64; i > 0; i>>=1)
    {
        if(x > val)
        {
            lower_pivot = pivot;
            lower = val;
            pivot+=i;
        }
        else
        {
            upper_pivot = pivot;
            upper = val;
            pivot-=i;
        }
        val = smem_code[pivot];
    }
    if(upper_pivot == 255)
        upper = smem_code[upper_pivot];
    if(lower_pivot == 0)
        lower = smem_code[lower_pivot];

    if(x > val)
    {
      float d2, d3;
      d2 = fabsf(x-val);
      d3 = fabsf(x-upper);
      if(d3 < d2)
        return upper_pivot;
      else
        return pivot;
    }
    else
    {
      float d1, d2;
      d1 = fabsf(x-lower);
      d2 = fabsf(x-val);
      if(d1 < d2)
        return lower_pivot;
      else
        return pivot;
    }
}

// the dynamic type goes from:
// values 0 to 32: 5.5e-07 to 0.011406250298023224
// boudary value: 0.0128125 
// values 32 to 64: 0.014218750409781933 to 0.10703125596046448
// boudary value:  0.1140625
// values 64 to 96: 0.12109375 to 0.5570312738418579
// boudary value:  0.5640625
// values 96 to 128: 0.5710937976837158 to 1.0
__device__ void quantize_atomic(float *temp, float* smem_code, float *x, int *out)
{
  int sign = 0;
  float absx = 0.0f;
  float err = 0.0f;
  int quadrant = 0;
  int lane_id = threadIdx.x % 32;
  int warp_id = threadIdx.x/32;

  // 1. determine quadrant
  // 2. compute errors
  // 3. find min error via atomics
  // 4. compare if local error is equal to min error
  // 5. write char index of min index

  for(int i = 0; i < 32; i++)
  {

    absx = x[i];
    sign = signbit(x[i]);
    if(lane_id == 0){ temp[warp_id] = -FLT_MAX; }
    __syncwarp();

    // 1. determine quadrant
    // 2. compute errors
    if(absx < 0.1140625)
    {
      if(absx > 0.0128125)
      {
        quadrant = 1;
        err = fabsf(absx-smem_code[128 + 32 + (threadIdx.x % 32)]);
      }
      else
      {
        quadrant = 0;
        err = fabsf(absx-smem_code[128 + (threadIdx.x % 32)]);
      }
    }
    else if(absx < 0.5640625)
    {
      quadrant = 2;
      err = fabsf(absx-smem_code[128 + 64 + (threadIdx.x % 32)]);
    }
    else
    {
      quadrant = 3;
      err = fabsf(absx-smem_code[128 + 96 + (threadIdx.x % 32)]);
    }

    // 3. find min error via atomics
    atomicMin(&temp[warp_id], err);
    __syncwarp();

    // 4. compare if local error is equal to min error
    if(err == temp[warp_id])
    {
      // 5. write char index of min index
      // 32 offset for each quadrant
      // signbit is 1 if negative
      out[i] = lane_id + (32*quadrant) + (sign == 0 ? 128 : 0);
    }
  }
}

__device__ unsigned char quantize_offset(float* smem_code, int lane, float x)
{
    int pivot = 127+lane-16;
    int upper_pivot = 255;
    int lower_pivot = 0;

    float val = smem_code[pivot];
    float lower = -1.0f;
    float upper = 1.0f;

    while((lower_pivot != pivot) && (upper_pivot != pivot))
    {
        if(x > val)
        {
            lower_pivot = pivot;
            lower = val;
            pivot = (upper_pivot+pivot) >> 1;
        }
        else
        {
            upper_pivot = pivot;
            upper = val;
            pivot = (lower_pivot+pivot) >> 1;
        }
        val = smem_code[pivot];
        //if((lower_pivot+1 == pivot) && (pivot+1 == upper_pivot)){ break; }
    }

    if(upper_pivot == 255)
        upper = smem_code[upper_pivot];
    if(lower_pivot == 0)
        lower = smem_code[lower_pivot];

    float d1, d2, d3;
    d1 = fabsf(x-lower);
    d2 = fabsf(x-val);
    d3 = fabsf(x-upper);

    if(d1 < d2) return lower_pivot;
    else if(d3 < d2) return upper_pivot;
    else return pivot;
}

__device__ unsigned char quantize_2D(float smem_code[][257], int id, float x)
{
    //bool is_positive = x > 0.0f;
    int pivot = x > 0.0f ? 191 : 63;
    int upper_pivot = x > 0.0f > 0.0 ? 255 : 127;
    int lower_pivot = x > 0.0f > 0.0 ? 127 : 0;

    float val = smem_code[id][pivot];
    float lower = -1.0f;
    float upper = 1.0f;

    // i>>=1 = {32, 16, 8, 4, 2, 1}
    for(int i = 32; i > 0; i>>=1)
    {
        if(x > val)
        {
            lower_pivot = pivot;
            lower = val;
            pivot+=i;
        }
        else
        {
            upper_pivot = pivot;
            upper = val;
            pivot-=i;
        }
        val = smem_code[id][pivot];
    }
    if(upper_pivot == 255 || upper_pivot == 127)
        upper = smem_code[id][upper_pivot];
    if(lower_pivot == 0 || lower_pivot == 127)
        lower = smem_code[id][lower_pivot];

    if(x > val)
    {
      float d2, d3;
      d2 = fabsf(x-val);
      d3 = fabsf(x-upper);
      if(d3 < d2)
        return upper_pivot;
      else
        return pivot;
    }
    else
    {
      float d1, d2;
      d1 = fabsf(x-lower);
      d2 = fabsf(x-val);
      if(d1 < d2)
        return lower_pivot;
      else
        return pivot;
    }
}

#define TH 1024
#define NUM 4
#define NUM_BLOCK 4096

template<typename T>
__launch_bounds__(TH, 1)
__global__ void kEstimateQuantiles(T *__restrict__ const A, float *code, const float offset, const T max_val, const int n)
{
  const int n_full = (NUM_BLOCK*(n/NUM_BLOCK)) + (n % NUM_BLOCK == 0 ? 0 : NUM_BLOCK);
  int valid_items = (blockIdx.x+1 == gridDim.x) ? n - (blockIdx.x*NUM_BLOCK) : NUM_BLOCK;
  const int base_idx = (blockIdx.x * NUM_BLOCK);
  const float reciprocal_num_blocks = 1.0f/(n < 4096 ? 1.0f : (n/NUM_BLOCK));

  T vals[NUM];

  typedef hipcub::BlockRadixSort<T, TH, NUM, hipcub::NullType, 4, true, hipcub::BLOCK_SCAN_RAKING> BlockRadixSort;
  typedef hipcub::BlockLoad<T, TH, NUM, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;

  __shared__ union {
      typename LoadFloat::TempStorage loadf;
      typename BlockRadixSort::TempStorage sort;
      int smem_qidx[NUM_BLOCK];
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_BLOCK)
  {
      valid_items = n - i > NUM_BLOCK ? NUM_BLOCK : n - i;

      // do not process half-blocks
      if(valid_items < NUM_BLOCK && n > NUM_BLOCK){ continue; }

      #pragma unroll 4
      for(int j = 0; j < NUM; j++)
          vals[j] = max_val;

      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(A[i]), vals, valid_items);

      #pragma unroll 4
      for(int j = 0; j < NUM; j++)
          vals[j] = ((float)vals[j]) * reciprocal_num_blocks;


      __syncthreads();
      // sort into striped pattern to mitigate bank conflicts
      // striped pattern index for thread 0 [0, 1024, 2048, 3096]
      // striped pattern index for thread 1 [1, 1025, 2049, 3097]
      BlockRadixSort(temp_storage.sort).SortBlockedToStriped(vals);

      __syncthreads();
      for(int j = threadIdx.x; j < NUM_BLOCK; j+=blockDim.x)
          temp_storage.smem_qidx[j] = -1;

      if(threadIdx.x < 256)
      {
          float q_interval = (1.0f-(2.0f*offset))/255.0f;
          int local_idx = round(((offset+(threadIdx.x*q_interval))*(valid_items-1)));
          temp_storage.smem_qidx[local_idx] = threadIdx.x;
      }

      __syncthreads();

      for(int i = threadIdx.x; i < NUM_BLOCK; i+=blockDim.x)
      {
          if(temp_storage.smem_qidx[i] != -1)
              atomicAdd(&code[temp_storage.smem_qidx[i]], vals[i/TH]);
      }
  }
}


__launch_bounds__(TH, 4)
__global__ void kQuantize(float * code, float * __restrict__ const A, unsigned char *out, const int n)
{
  const int n_full = (NUM_BLOCK*(n/NUM_BLOCK)) + (n % NUM_BLOCK == 0 ? 0 : NUM_BLOCK);
  int valid_items = (blockIdx.x+1 == gridDim.x) ? n - (blockIdx.x*NUM_BLOCK) : NUM_BLOCK;
  const int base_idx = (blockIdx.x * NUM_BLOCK);

  float vals[NUM];
  unsigned char qvals[NUM];
  //const int lane_id = threadIdx.x % 2;

  typedef hipcub::BlockLoad<float, TH, NUM, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockStore<unsigned char, TH, NUM, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;


  __shared__ typename LoadFloat::TempStorage loadf;
  __shared__ typename StoreChar::TempStorage storec;
  __shared__ float smem_code[256];
  //__shared__ float smem_code[2][257];

  if(threadIdx.x < 256)
  {
    smem_code[threadIdx.x] = code[threadIdx.x];
    //smem_code[0][threadIdx.x] = code[threadIdx.x];
    //smem_code[1][threadIdx.x] = smem_code[0][threadIdx.x];
  }


  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_BLOCK)
  {
      valid_items = n - i > NUM_BLOCK ? NUM_BLOCK : n - i;

      __syncthreads();
      LoadFloat(loadf).Load(&(A[i]), vals, valid_items);

      #pragma unroll 4
      for(int j = 0; j < NUM; j++)
        qvals[j] = quantize(smem_code, vals[j]);
        //qvals[j] = quantize_2D(smem_code, lane_id, vals[j]);

      __syncthreads();
      StoreChar(storec).Store(&(out[i]), qvals, valid_items);
  }
}

template<typename T, int BLOCK_SIZE, int NUM_PER_TH>
__launch_bounds__(TH, 4)
__global__ void kQuantizeBlockwise(float * code, T * __restrict__ const A, float *absmax, unsigned char *out, const int n)
{
  const int n_full = gridDim.x * BLOCK_SIZE;
  int valid_items = 0;
  const int base_idx = (blockIdx.x * BLOCK_SIZE);

  T vals[NUM];
  unsigned char qvals[NUM];
  float local_abs_max = -FLT_MAX;

  typedef hipcub::BlockLoad<T, BLOCK_SIZE/NUM_PER_TH, NUM_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
  typedef hipcub::BlockStore<unsigned char, BLOCK_SIZE/NUM_PER_TH, NUM_PER_TH, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;
  typedef hipcub::BlockReduce<float, BLOCK_SIZE/NUM_PER_TH> BlockReduce;

  __shared__ typename LoadT::TempStorage loadt;
  __shared__ typename StoreChar::TempStorage storec;
  __shared__ typename BlockReduce::TempStorage reduce;
  __shared__ float smem_code[256];
  __shared__ float smem_absmax_value[1];

  if(threadIdx.x < 256)
    smem_code[threadIdx.x] = code[threadIdx.x];

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
  {
      valid_items = n - i > BLOCK_SIZE ? BLOCK_SIZE : n - i;
      local_abs_max = -FLT_MAX;

      __syncthreads();
      LoadT(loadt).Load(&(A[i]), vals, valid_items, (T)0.0f);

    // 1. compute local max
    // 2. broadcast local max
    // 3. normalize inputs and quantize

     #pragma unroll NUM_PER_TH
     for(int j = 0; j < NUM_PER_TH; j++)
        local_abs_max = fmaxf(local_abs_max, fabsf((float)vals[j]));

     local_abs_max = BlockReduce(reduce).Reduce(local_abs_max, hipcub::Max(), valid_items);


     if(threadIdx.x == 0)
       smem_absmax_value[0] = local_abs_max;

     __syncthreads();

     if(threadIdx.x == 0)
       absmax[i/BLOCK_SIZE] = local_abs_max;
     else
       local_abs_max = smem_absmax_value[0];

     __syncwarp();

     local_abs_max = 1.0f/local_abs_max;

     #pragma unroll NUM_PER_TH
     for(int j = 0; j < NUM_PER_TH; j++)
        qvals[j] = quantize(smem_code, ((float)vals[j])*local_abs_max);

     __syncthreads();
     StoreChar(storec).Store(&(out[i]), qvals, valid_items);
  }
}

template<typename T, int BLOCK_SIZE, int THREADS, int NUM_PER_TH>
__global__ void kDequantizeBlockwise(float *code, unsigned char * __restrict__ const A, float * __restrict__ const absmax, T *out, const int n)
{

  const int n_full = gridDim.x * BLOCK_SIZE;
  int valid_items = 0;
  const int base_idx = (blockIdx.x * BLOCK_SIZE);

  T vals[NUM];
  unsigned char qvals[NUM];
  float local_abs_max = -FLT_MAX;

  typedef hipcub::BlockLoad<unsigned char, THREADS, NUM_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;
  typedef hipcub::BlockStore<T, THREADS, NUM_PER_TH, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

  __shared__ typename LoadChar::TempStorage loadchar;
  __shared__ typename StoreT::TempStorage storet;
  __shared__ float smem_code[256];

  if(threadIdx.x < 256)
    smem_code[threadIdx.x] = code[threadIdx.x];

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
  {
      valid_items = n - i > BLOCK_SIZE ? BLOCK_SIZE : n - i;
      local_abs_max = absmax[i/BLOCK_SIZE];

      __syncthreads();
      LoadChar(loadchar).Load(&(A[i]), qvals, valid_items, 128);

      #pragma unroll NUM_PER_TH
      for(int j = 0; j < NUM_PER_TH; j++)
        vals[j] = smem_code[qvals[j]]*local_abs_max;

      __syncthreads();
      StoreT(storet).Store(&(out[i]), vals, valid_items);
  }
}


__global__ void kDequantize(float *code, unsigned char *A, float *out, const int n)
{
	const unsigned int numThreads = blockDim.x * gridDim.x;
	const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	__shared__ float smem_code[256];
	if(threadIdx.x < 256)
	{
		smem_code[threadIdx.x] = code[threadIdx.x];
	}

	__syncthreads();

	for (int i = idx;i < n; i += numThreads)
	{
		out[i] = smem_code[A[i]];
	}
}



template<typename T, int OPTIMIZER, int BLOCK_SIZE, int NUM_VALS>
__launch_bounds__(BLOCK_SIZE/NUM_VALS, 1)
__global__ void kPreconditionOptimizer32bit2State(T* g, T* p, 
                float* state1, float* state2, float *unorm,
                const float beta1, const float beta2, const float eps, const float weight_decay,
                const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n)
{

  const int n_full = (BLOCK_SIZE*(n/BLOCK_SIZE)) + (n % BLOCK_SIZE == 0 ? 0 : BLOCK_SIZE);
  const int base_idx = (blockIdx.x * blockDim.x * NUM_VALS);
  int valid_items = 0;

  T g_vals[NUM_VALS];

  float s1_vals[NUM_VALS];
  float s2_vals[NUM_VALS];

  const float correction1 = 1.0f/(1.0f - powf(beta1, step));
  const float correction2 = 1.0f/(1.0f - powf(beta2, step));

  typedef hipcub::BlockLoad<T, BLOCK_SIZE/NUM_VALS, NUM_VALS, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> Load;
  typedef hipcub::BlockLoad<float, BLOCK_SIZE/NUM_VALS, NUM_VALS, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockReduce<float, BLOCK_SIZE/NUM_VALS> BlockReduce;

  __shared__ union {
      typename Load::TempStorage load;
      typename LoadFloat::TempStorage loadf;
      typename BlockReduce::TempStorage reduce;
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
  {
      valid_items = n - i >= (BLOCK_SIZE) ? (BLOCK_SIZE) : n - i;

      __syncthreads();
      Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items, 0.0f);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items, 0.0f);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state2[i]), s2_vals, valid_items, 0.0f);

      # pragma unroll NUM_VALS
      for(unsigned int j = 0; j < NUM_VALS; j++)
        g_vals[j] = gnorm_scale*((float)g_vals[j]);

      # pragma unroll NUM_VALS
      for(unsigned int j = 0; j < NUM_VALS; j++)
      {
          switch(OPTIMIZER)
          {
              case ADAM: 
                  if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                  {
                    s1_vals[j] = s1_vals[j]*beta1 + ((1.0f -beta1)*((float)g_vals[j]));
                    s2_vals[j] = s2_vals[j]*beta2 + ((1.0f -beta2)*(((float)g_vals[j])*((float)g_vals[j])));
                    s1_vals[j] *= correction1;
                    s2_vals[j] *= correction2;
                    s1_vals[j] = s1_vals[j]/(sqrtf(s2_vals[j])+eps); // update
                    s1_vals[j] *= s1_vals[j]; // update l2 norm (update*update)
                  }
                  break;
          }
      }

      # pragma unroll NUM_VALS-1
      for(unsigned int j = 1; j < NUM_VALS; j++)
          s1_vals[0] += s1_vals[j];

      __syncthreads();
      s1_vals[0] = BlockReduce(temp_storage.reduce).Sum(s1_vals[0]);

      if(threadIdx.x == 0)
        atomicAdd(&unorm[0], s1_vals[0]);

      __syncwarp();
  }
}



#define NUM_PER_THREAD 4

template<typename T, int OPTIMIZER>
__launch_bounds__(TH, 1)
__global__ void kOptimizer32bit2State(T* g, T* p, 
                float* state1, float* state2, float *unorm, const float max_unorm, const float param_norm,
                const float beta1, const float beta2, const float eps, const float weight_decay,
                const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n)
{

  const int n_full = ((TH*NUM_PER_THREAD)*(n/(TH*NUM_PER_THREAD))) + (n % (TH*NUM_PER_THREAD) == 0 ? 0 : (TH*NUM_PER_THREAD));
  const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
  int valid_items = 0;
  float update_scale = 0.0f;
  T g_vals[NUM_PER_THREAD];
  T p_vals[NUM_PER_THREAD];

  float s1_vals[NUM_PER_THREAD];
  float s2_vals[NUM_PER_THREAD];

  const float correction1 = 1.0f - powf(beta1, step);
  const float correction2 = sqrtf(1.0f - powf(beta2, step));
  const float step_size = -lr*correction2/correction1;

  if(max_unorm > 0.0f)
  {
    update_scale = max_unorm > 0.0f ? sqrtf(unorm[0]) : 1.0f;
    if(update_scale > max_unorm*param_norm){ update_scale = (max_unorm*param_norm)/update_scale; }
    else{ update_scale = 1.0f; }
  }
  else{ update_scale = 1.0f; }

  typedef hipcub::BlockLoad<T, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> Load;
  typedef hipcub::BlockStore<T, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> Store;

  typedef hipcub::BlockLoad<float, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockStore<float, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreFloat;

  __shared__ union {
      typename Load::TempStorage load;
      typename Store::TempStorage store;
      typename LoadFloat::TempStorage loadf;
      typename StoreFloat::TempStorage storef;
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*TH*NUM_PER_THREAD)
  {
      valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

      __syncthreads();
      Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state2[i]), s2_vals, valid_items);
      __syncthreads();
      Load(temp_storage.load).Load(&(p[i]), p_vals, valid_items);

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
        g_vals[j] = gnorm_scale*((float)g_vals[j]);

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
      {
          switch(OPTIMIZER)
          {
              case ADAM: 
                  if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                  {
                    s1_vals[j] = s1_vals[j]*beta1 + ((1.0f -beta1)*((float)g_vals[j]));
                    s2_vals[j] = s2_vals[j]*beta2 + ((1.0f -beta2)*(((float)g_vals[j])*((float)g_vals[j])));
                    p_vals[j] = ((float)p_vals[j]) + (update_scale*step_size*(s1_vals[j]/(sqrtf(s2_vals[j])+(eps*correction2))));
                  }
                  break;
          }
      }

      __syncthreads();
      Store(temp_storage.store).Store(&(p[i]), p_vals, valid_items);
      __syncthreads();
      StoreFloat(temp_storage.storef).Store(&(state1[i]), s1_vals, valid_items);
      __syncthreads();
      StoreFloat(temp_storage.storef).Store(&(state2[i]), s2_vals, valid_items);
  }
}

template<typename T, int OPTIMIZER, int BLOCK_SIZE, int NUM_VALS>
__launch_bounds__(TH, 1)
__global__ void kPreconditionOptimizer32bit1State(T* g, T* p, 
                float* state1, float *unorm,
                const float beta1, const float eps, const float weight_decay,
                const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n)
{

  const int n_full = (BLOCK_SIZE*(n/BLOCK_SIZE)) + (n % BLOCK_SIZE == 0 ? 0 : BLOCK_SIZE);
  const int base_idx = (blockIdx.x * blockDim.x * NUM_VALS);
  int valid_items = 0;

  T g_vals[NUM_VALS];

  float s1_vals[NUM_VALS];

  typedef hipcub::BlockLoad<T, BLOCK_SIZE/NUM_VALS, NUM_VALS, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> Load;
  typedef hipcub::BlockLoad<float, BLOCK_SIZE/NUM_VALS, NUM_VALS, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockReduce<float, BLOCK_SIZE/NUM_VALS> BlockReduce;

  __shared__ union {
      typename Load::TempStorage load;
      typename LoadFloat::TempStorage loadf;
      typename BlockReduce::TempStorage reduce;
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
  {
      valid_items = n - i >= (BLOCK_SIZE) ? (BLOCK_SIZE) : n - i;

      __syncthreads();
      Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items, 0.0f);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items, 0.0f);

      # pragma unroll NUM_VALS
      for(unsigned int j = 0; j < NUM_VALS; j++)
        g_vals[j] = gnorm_scale*((float)g_vals[j]);

      # pragma unroll NUM_VALS
      for(unsigned int j = 0; j < NUM_VALS; j++)
      {
          switch(OPTIMIZER)
          {
              case MOMENTUM: 
                  if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                  {
                    if(step == 1)
                      s1_vals[j] = (float)g_vals[j]; // state update
                    else
                      s1_vals[j] = s1_vals[j]*beta1 + ((float)g_vals[j]); // state update
                    s1_vals[j] = s1_vals[j]*s1_vals[j]; // update norm
                  }
                  break;
              case RMSPROP: 
                  if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                  {
                    s1_vals[j] = s1_vals[j]*beta1 + ((1.0f-beta1)*((float)g_vals[j])*((float)g_vals[j])); // state update
                    s1_vals[j] = __fdividef((float)g_vals[j],sqrtf(s1_vals[j])+eps); // update value
                    s1_vals[j] = s1_vals[j]*s1_vals[j]; // update norm
                  }
                  break;
          }
      }

      # pragma unroll
      for(unsigned int j = 1; j < NUM_VALS; j++)
        s1_vals[0] += s1_vals[j];

      __syncthreads();
      s1_vals[0] = BlockReduce(temp_storage.reduce).Sum(s1_vals[0], valid_items);

      if(threadIdx.x == 0)
        atomicAdd(&unorm[0], s1_vals[0]);

      __syncwarp();
  }
}

template<typename T, int OPTIMIZER>
__launch_bounds__(TH, 1)
__global__ void kOptimizer32bit1State(T *g, T *p, 
                float *state1, float *unorm, const float max_unorm, const float param_norm,
                const float beta1, const float eps, const float weight_decay,
                const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n)
{

  const int n_full = ((TH*NUM_PER_THREAD)*(n/(TH*NUM_PER_THREAD))) + (n % (TH*NUM_PER_THREAD) == 0 ? 0 : (TH*NUM_PER_THREAD));
  const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
  int valid_items = 0;
  float update_scale = 0.0f;

  if(max_unorm > 0.0f)
  {
    update_scale = max_unorm > 0.0f ? sqrtf(unorm[0]) : 1.0f;
    if(update_scale > max_unorm*param_norm+eps){ update_scale = (max_unorm*param_norm+eps)/update_scale; }
    else{ update_scale = 1.0f; }
  }
  else{ update_scale = 1.0f; }

  T g_vals[NUM_PER_THREAD];
  T p_vals[NUM_PER_THREAD];

  float s1_vals[NUM_PER_THREAD];

  typedef hipcub::BlockLoad<T, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> Load;
  typedef hipcub::BlockStore<T, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> Store;

  typedef hipcub::BlockLoad<float, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockStore<float, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreFloat;

  __shared__ union {
      typename Load::TempStorage load;
      typename Store::TempStorage store;
      typename LoadFloat::TempStorage loadf;
      typename StoreFloat::TempStorage storef;
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*TH*NUM_PER_THREAD)
  {
      valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

      __syncthreads();
      Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items);
      __syncthreads();
      Load(temp_storage.load).Load(&(p[i]), p_vals, valid_items);

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
        g_vals[j] = gnorm_scale*((float)g_vals[j]);

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
      {
          switch(OPTIMIZER)
          {
              case MOMENTUM: 
                  if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                  {
                    if(step == 1)
                      s1_vals[j] = (float)g_vals[j];
                    else
                      s1_vals[j] = s1_vals[j]*beta1 + ((float)g_vals[j]);

                    p_vals[j] = ((float)p_vals[j]) + update_scale*(-lr*(s1_vals[j]));
                  }
                  break;
              case RMSPROP: 
                  if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                  {
                    s1_vals[j] = s1_vals[j]*beta1 + ((1.0f-beta1)*((float)g_vals[j])*((float)g_vals[j]));
                    p_vals[j] = ((float)p_vals[j]) - update_scale*(lr*__fdividef((float)g_vals[j],sqrtf((float)s1_vals[j])+eps));
                  }
                  break;
          }
      }

      __syncthreads();
      Store(temp_storage.store).Store(&(p[i]), p_vals, valid_items);
      __syncthreads();
      StoreFloat(temp_storage.storef).Store(&(state1[i]), s1_vals, valid_items);
  }
}


#define NUM8BIT 16
#define NUM_THREADS 256
#define NUM_PER_BLOCK 4096

template<typename T, int OPTIMIZER>
__global__ void
__launch_bounds__(NUM_THREADS, 2)
kPreconditionOptimizerStatic8bit2State(T* p, T* __restrict__ const g, unsigned char*__restrict__  const state1, unsigned char* __restrict__ const state2,
                float *unorm,
                const float beta1, const float beta2,
                const float eps, const int step,
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                const float gnorm_scale, const int n)
{
    const int n_full = gridDim.x * NUM_PER_BLOCK;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
    int valid_items = n - (blockIdx.x*NUM_PER_BLOCK) > NUM_PER_BLOCK ? NUM_PER_BLOCK : n - (blockIdx.x*NUM_PER_BLOCK);
    float g_val = 0.0f;
    float local_max_s1 = -FLT_MAX;
    float local_max_s2 = -FLT_MAX;
    float local_unorm = 0.0f;

    float s2_vals[NUM8BIT];
    float s1_vals[NUM8BIT];
    T g_vals[NUM8BIT];
    unsigned char m_c1[NUM8BIT];
    unsigned char r_c2[NUM8BIT];

    typedef hipcub::BlockRadixSort<float, NUM_THREADS, NUM8BIT, hipcub::NullType, 6, true, hipcub::BLOCK_SCAN_RAKING> BlockRadixSort;
    typedef hipcub::BlockLoad<T, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadUInt8;
    typedef hipcub::BlockReduce<float, NUM_THREADS> BlockReduce;


    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadUInt8::TempStorage loadc;
        typename BlockRadixSort::TempStorage sort;
        typename BlockReduce::TempStorage reduce;
    } temp_storage;

    __shared__ float smem_quantiles1[256];
    __shared__ float smem_quantiles2[256];

    if(threadIdx.x < 256)
    {
        smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];
        smem_quantiles2[threadIdx.x] = quantiles2[threadIdx.x];
    }

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += NUM_THREADS*gridDim.x*NUM8BIT)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadUInt8(temp_storage.loadc).Load(&(state1[i]), m_c1, valid_items, 128);
        __syncthreads();
        LoadUInt8(temp_storage.loadc).Load(&(state2[i]), r_c2, valid_items, 128);
        __syncthreads();

        #pragma unroll 16
        for(int j = 0; j < NUM8BIT; j++)
        {
            g_val = g_vals[j];
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[m_c1[j]]*max1[0]*beta1;
            s1_vals[j] += (1.0f-beta1)*g_val;
            local_max_s1 = fmaxf(local_max_s1, fabsf(s1_vals[j]));
        }

        #pragma unroll 16
        for(int j = 0; j < NUM8BIT; j++)
        {
            g_val = g_vals[j];
            g_val *= gnorm_scale;
            s2_vals[j] = smem_quantiles2[r_c2[j]]*max2[0]*beta2;
            s2_vals[j] += (1.0f-beta2)*g_val*g_val;
            local_max_s2 = fmaxf(local_max_s2, fabsf(s2_vals[j]));
        }

        if(unorm != NULL)
        {
          #pragma unroll 16
          for(int j = 0; j < NUM8BIT; j++)
          {
            float correction1 = __fdividef(1.0f, 1.0f - powf(beta1, step));
            float correction2 = __fdividef(1.0f, 1.0f - powf(beta2, step));
            s1_vals[j] *= correction1;
            s2_vals[j] *= correction2;
            float update_val = s1_vals[j]/(sqrtf(s2_vals[j])+eps); // update
            local_unorm += update_val*update_val;
          }
        }
    }

    __syncthreads();
    local_max_s1 = BlockReduce(temp_storage.reduce).Reduce(local_max_s1, hipcub::Max(), valid_items);
    __syncthreads();
    local_max_s2 = BlockReduce(temp_storage.reduce).Reduce(local_max_s2, hipcub::Max(), valid_items);
    if(unorm != NULL)
    {
      __syncthreads();
      local_unorm = BlockReduce(temp_storage.reduce).Reduce(local_unorm, hipcub::Sum(), valid_items);
    }

    if(threadIdx.x == 0)
    {
        atomicMax(&new_max1[0], local_max_s1);
        atomicMax(&new_max2[0], local_max_s2);
        if(unorm != NULL){ atomicAdd(&unorm[0], local_unorm); }
    }
}

#define NUM_PER_THREAD2 4
#define NUM_THREADS2 1024
#define NUM_PER_BLOCK2 4096

template<typename T, int OPTIMIZER>
__global__ void
kOptimizerStatic8bit2State(T* p, T* const g, unsigned char* state1, unsigned char* state2,
                const float *unorm, const float max_unorm, const float param_norm, \
                const float beta1, const float beta2,
                const float eps, const int step, const float lr,
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                float weight_decay,
                const float gnorm_scale, const int n)
{

    const int n_full = (blockDim.x * gridDim.x)*NUM_PER_THREAD2;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD2);
    int valid_items = 0;
    float g_val = 0.0f;
    float s1_vals[NUM_PER_THREAD2];
    float s2_vals[NUM_PER_THREAD2];
    const float correction1 = 1.0f - powf(beta1, step);
    const float correction2 = sqrtf(1.0f - powf(beta2, step));
    const float step_size = -lr*correction2/correction1;
    //const float step_size = -lr*correction2/correction1;
    float new_max_val1 = 1.0f/new_max1[0];
    float new_max_val2 = 1.0f/new_max2[0];
    float update_scale = 1.0f;

    if(max_unorm > 0.0f)
    {
      update_scale = max_unorm > 0.0f ? sqrtf(unorm[0]) : 1.0f;
      if(update_scale > max_unorm*param_norm){ update_scale = (max_unorm*param_norm)/update_scale; }
      else{ update_scale = 1.0f; }
    }
    else{ update_scale = 1.0f; }

    unsigned char c1s[NUM_PER_THREAD2];
    unsigned char c2s[NUM_PER_THREAD2];
    T p_vals[NUM_PER_THREAD2];
    T g_vals[NUM_PER_THREAD2];
    typedef hipcub::BlockLoad<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;

    typedef hipcub::BlockStore<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;
    typedef hipcub::BlockStore<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

    __shared__ float smem_quantiles1[256];
    __shared__ float smem_quantiles2[256];

    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadChar::TempStorage loadc;
        typename StoreChar::TempStorage storec;
        typename StoreT::TempStorage storeh;
    } temp_storage;

    if(threadIdx.x < 512)
    {
        if(threadIdx.x < 256)
            smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];
        else
            smem_quantiles2[threadIdx.x-256] = quantiles2[threadIdx.x-256];
    }

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_THREADS2*NUM_PER_THREAD2)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state1[i]), c1s, valid_items, 128);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state2[i]), c2s, valid_items, 128);
        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(p[i]), p_vals, valid_items);

        if((i + (threadIdx.x*NUM_PER_THREAD2) + NUM_PER_THREAD2) > n){ continue; }

        # pragma unroll 4
        for(unsigned int j = 0; j < NUM_PER_THREAD2; j++)
        {
            g_val = float(g_vals[j]);
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[c1s[j]];
            s1_vals[j] = s1_vals[j]*max1[0];

            s1_vals[j] = (s1_vals[j]*beta1) + (((1.0f-beta1)*g_val));

            c1s[j] = quantize(smem_quantiles1, s1_vals[j]*new_max_val1);

            // make sure state1 term has still the same sign after quantization
            // (not needed for state2 term which has only positive values)
            if(signbit(smem_quantiles1[c1s[j]]) != signbit(s1_vals[j]))
            {
              if(s1_vals[j] > 0.0f)
                  c1s[j] += 1;
              else
                  c1s[j] -= 1;
            }

            s2_vals[j] = smem_quantiles2[c2s[j]];
            s2_vals[j] = s2_vals[j]*max2[0];
            s2_vals[j] = (s2_vals[j]*beta2) + (((1.0f-beta2)*g_val*g_val));
            c2s[j] = quantize(smem_quantiles2, s2_vals[j]*new_max_val2);
        }

        # pragma unroll 4
        for(unsigned int j = 0; j < NUM_PER_THREAD2; j++)
        {
            p_vals[j] = (T)(((float)p_vals[j]) + ((update_scale*step_size*(s1_vals[j]/(sqrtf(s2_vals[j])+(correction2*eps))))));
            if(weight_decay > 0.0f)
                p_vals[j] = ((float)p_vals[j])*(1.0f-(lr*weight_decay));
        }

        StoreT(temp_storage.storeh).Store(&(p[i]), p_vals, valid_items);
        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state1[i]), c1s, valid_items);
        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state2[i]), c2s, valid_items);
        __syncthreads();
    }
}


template<typename T, int OPTIMIZER>
__global__ void
__launch_bounds__(NUM_THREADS, 2)
kPreconditionOptimizerStatic8bit1State(T* p, T* __restrict__ const g, unsigned char*__restrict__  const state1, 
                float *unorm,
                const float beta1, 
                const float eps, const int step,
                float* __restrict__ const quantiles1, 
                float* max1, float* new_max1, 
                const float gnorm_scale, const int n)
{
    const int n_full = gridDim.x * NUM_PER_BLOCK;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
    int valid_items = n - (blockIdx.x*NUM_PER_BLOCK) > NUM_PER_BLOCK ? NUM_PER_BLOCK : n - (blockIdx.x*NUM_PER_BLOCK);
    float g_val = 0.0f;
    float local_max_s1 = -FLT_MAX;
    float local_unorm = 0.0f;

    float s1_vals[NUM8BIT];
    T g_vals[NUM8BIT];
    unsigned char m_c1[NUM8BIT];

    typedef hipcub::BlockRadixSort<float, NUM_THREADS, NUM8BIT, hipcub::NullType, 6, true, hipcub::BLOCK_SCAN_RAKING> BlockRadixSort;
    typedef hipcub::BlockLoad<T, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadUInt8;
    typedef hipcub::BlockReduce<float, NUM_THREADS> BlockReduce;


    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadUInt8::TempStorage loadc;
        typename BlockRadixSort::TempStorage sort;
        typename BlockReduce::TempStorage reduce;
    } temp_storage;

    __shared__ float smem_quantiles1[256];

    if(threadIdx.x < 256)
      smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_THREADS*NUM8BIT)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadUInt8(temp_storage.loadc).Load(&(state1[i]), m_c1, valid_items, 128);

        #pragma unroll 16
        for(int j = 0; j < NUM8BIT; j++)
        {
            g_val = g_vals[j];
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[m_c1[j]]*max1[0];
            switch(OPTIMIZER)
            {
                case MOMENTUM: 
                    //TODO: if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                    //{
                      if(step == 1)
                        s1_vals[j] = (float)g_vals[j];
                      else
                        s1_vals[j] = s1_vals[j]*beta1 + ((float)g_vals[j]);
                    //}
                    if(unorm != NULL)
                      local_unorm += s1_vals[j]*s1_vals[j];
                    break;
              case RMSPROP: 
                    s1_vals[j] = s1_vals[j]*beta1 + ((1.0f-beta1)*(g_val*g_val));
                  break;
            }

            local_max_s1 = fmaxf(local_max_s1, fabsf(s1_vals[j]));
        }
    }

    __syncthreads();
    local_max_s1 = BlockReduce(temp_storage.reduce).Reduce(local_max_s1, hipcub::Max(), valid_items);
    if(threadIdx.x == 0){ atomicMax(&new_max1[0], local_max_s1); }
    if(unorm != NULL)
    {
      __syncthreads();
      local_unorm = BlockReduce(temp_storage.reduce).Reduce(local_unorm, hipcub::Sum(), valid_items);
      if(threadIdx.x == 0){ atomicAdd(&unorm[0], local_unorm); }
    }

}

template<typename T, int OPTIMIZER>
__global__ void
kOptimizerStatic8bit1State(T* p, T* const g, unsigned char* state1,
                const float *unorm, const float max_unorm, const float param_norm,
                const float beta1, 
                const float eps, const int step, const float lr,
                float* __restrict__ const quantiles1, 
                float* max1, float* new_max1, 
                float weight_decay,
                const float gnorm_scale, const int n)
{

    const int n_full = (blockDim.x * gridDim.x)*NUM_PER_THREAD2;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD2);
    int valid_items = 0;
    float g_val = 0.0f;
    float s1_vals[NUM_PER_THREAD2];
    float new_max_val1 = 1.0f/new_max1[0];
    float update_scale = 1.0f;

    if(max_unorm > 0.0f)
    {
      update_scale = max_unorm > 0.0f ? sqrtf(unorm[0]) : 1.0f;
      if(update_scale > max_unorm*param_norm){ update_scale = (max_unorm*param_norm)/update_scale; }
      else{ update_scale = 1.0f; }
    }
    else{ update_scale = 1.0f; }

    unsigned char c1s[NUM_PER_THREAD2];
    T p_vals[NUM_PER_THREAD2];
    T g_vals[NUM_PER_THREAD2];
    typedef hipcub::BlockLoad<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;

    typedef hipcub::BlockStore<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;
    typedef hipcub::BlockStore<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

    __shared__ float smem_quantiles1[256];

    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadChar::TempStorage loadc;
        typename StoreChar::TempStorage storec;
        typename StoreT::TempStorage storeh;
    } temp_storage;

    if(threadIdx.x < 256)
        smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_THREADS2*NUM_PER_THREAD2)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state1[i]), c1s, valid_items, 128);
        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(p[i]), p_vals, valid_items);

        if((i + (threadIdx.x*NUM_PER_THREAD2) + NUM_PER_THREAD2) > n){ continue; }

        # pragma unroll 4
        for(unsigned int j = 0; j < NUM_PER_THREAD2; j++)
        {
            g_val = float(g_vals[j]);
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[c1s[j]]*max1[0];

            switch(OPTIMIZER)
            {
                case MOMENTUM: 
                  //TODO: if(!is_sparse || ((float)g_vals[j] != 0.0f && is_sparse))
                  //{
                    if(step == 1)
                      s1_vals[j] = g_vals[j];
                    else
                      s1_vals[j] = s1_vals[j]*beta1 + ((float)g_vals[j]);

                    //TODO: if(weight_decay > 0.0f)
                    p_vals[j] = ((float)p_vals[j]) + (-lr*update_scale*(s1_vals[j]));
                  //}
                  break;
              case RMSPROP: 
                  s1_vals[j] = s1_vals[j]*beta1 + ((1.0f-beta1)*(g_val*g_val));
                  p_vals[j] = ((float)p_vals[j]) - (lr*__fdividef(g_val,sqrtf(s1_vals[j])+eps));
                  break;
            }

            c1s[j] = quantize(smem_quantiles1, s1_vals[j]*new_max_val1);

            // make sure state1 term has still the same sign after quantization
            if(signbit(smem_quantiles1[c1s[j]]) != signbit(s1_vals[j]))
            {
              if(s1_vals[j] > 0.0f)
                  c1s[j] += 1;
              else
                  c1s[j] -= 1;
            }
        }

        StoreT(temp_storage.storeh).Store(&(p[i]), p_vals, valid_items);
        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state1[i]), c1s, valid_items);
        __syncthreads();
    }
}


template<typename T, int BLOCK_SIZE, int NUM_VALS>
__global__ void kPercentileClipping(T * __restrict__ g, float *gnorm_vec, int step, const int n)
{
  const int n_full = (BLOCK_SIZE*(n/BLOCK_SIZE)) + (n % BLOCK_SIZE == 0 ? 0 : BLOCK_SIZE);
  int valid_items = 0;

  typedef hipcub::BlockReduce<float, BLOCK_SIZE/NUM_VALS> BlockReduce;
  typedef hipcub::BlockLoad<T, BLOCK_SIZE/NUM_VALS, NUM_VALS, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;

  __shared__ typename BlockReduce::TempStorage reduce;

  __shared__ typename LoadT::TempStorage loadT;
  T vals[NUM_VALS];
  float local_sum = 0.0f;

  for (unsigned int i = (blockIdx.x * BLOCK_SIZE); i < n_full; i += gridDim.x*BLOCK_SIZE)
  {
      valid_items = n - i > BLOCK_SIZE ? BLOCK_SIZE : n - i;
      local_sum = 0.0f;

      __syncthreads();
      LoadT(loadT).Load(&(g[i]), vals, valid_items, (T)0.0f);

     #pragma unroll NUM_VALS
     for(int j = 0; j < NUM_VALS; j++)
       local_sum += ((float)vals[j])*((float)vals[j]);

    local_sum = BlockReduce(reduce).Sum(local_sum, valid_items);
    if(threadIdx.x == 0)
    {
      if(step == 1)
      {
        // initialize with the same norm for all positions
        //#pragma unroll 10
        for(int j = 0; j < 100; j++)
          atomicAdd(&gnorm_vec[j], local_sum);
      }
      else
          atomicAdd(&gnorm_vec[step % 100], local_sum);
    }

  }
}


#define LANES 2
template<typename T, int OPTIMIZER, int BLOCK_SIZE, int N_PER_TH>
__launch_bounds__(256, 3)
__global__ void
kOptimizerStatic8bit2StateBlockwise(T* p, T* __restrict__ const g, unsigned char* state1, unsigned char* state2,
                const float beta1, const float beta2,
                const float eps, const int step, const float lr,
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2,
                float* absmax1, float* absmax2, 
                float weight_decay,
                const float gnorm_scale, const int n)
{

    //const int n_full = n + (n%BLOCK_SIZE);
    const int n_full = gridDim.x * BLOCK_SIZE;
    const int base_idx = (blockIdx.x * BLOCK_SIZE);
    int valid_items = 0;
    float g_val = 0.0f;
    float s1_vals[N_PER_TH];
    float s2_vals[N_PER_TH];
    // 2-5%
    const float correction1 = 1.0f - powf(beta1, step);
    const float correction2 = sqrtf(1.0f - powf(beta2, step));
    const float step_size = -lr*correction2/correction1;
    const int lane_id = threadIdx.x % LANES;
    float new_local_abs_max1 = -FLT_MAX;
    float new_local_abs_max2 = -FLT_MAX;

    unsigned char c1s[N_PER_TH];
    unsigned char c2s[N_PER_TH];
    //T p_vals[N_PER_TH];
    T g_vals[N_PER_TH];
    typedef hipcub::BlockLoad<T, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;

    typedef hipcub::BlockStore<unsigned char, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;
    typedef hipcub::BlockStore<T, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

    //__shared__ float smem_quantiles1[256];
    //__shared__ float smem_quantiles2[256];
    __shared__ float smem_quantiles1[LANES][257];
    __shared__ float smem_quantiles2[LANES][257];
    typedef hipcub::BlockReduce<float, BLOCK_SIZE/N_PER_TH> BlockReduce1;
    typedef hipcub::BlockReduce<float, BLOCK_SIZE/N_PER_TH> BlockReduce2;
    __shared__ typename BlockReduce1::TempStorage reduce1;
    __shared__ typename BlockReduce2::TempStorage reduce2;
    __shared__ float smem_exchange1[1];
    __shared__ float smem_exchange2[1];

    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadChar::TempStorage loadc;
        typename StoreChar::TempStorage storec;
        typename StoreT::TempStorage storeh;
        //float quantiles[8][257];
    } temp_storage;
    // init: 0.2 -> 0.23

    // 0.23 -> 0.23
    if(BLOCK_SIZE/N_PER_TH >= 512)
    {
      //if(threadIdx.x < 512)
      //{
      //    if(threadIdx.x < 256)
      //        smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];
      //    else
      //        smem_quantiles2[threadIdx.x-256] = quantiles2[threadIdx.x-256];
      //}
    }
    else
    {
        //smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];
        //smem_quantiles2[threadIdx.x] = quantiles2[threadIdx.x];
        smem_quantiles1[0][threadIdx.x] = quantiles1[threadIdx.x];
        smem_quantiles2[0][threadIdx.x] = quantiles2[threadIdx.x];
        # pragma unroll
        for(unsigned int j = 1; j < LANES; j++)
        {
          smem_quantiles1[j][threadIdx.x] = smem_quantiles1[0][threadIdx.x];
          smem_quantiles2[j][threadIdx.x] = smem_quantiles2[0][threadIdx.x];
        }
    }

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
    {
        // loads: 0.23 -> 0.85/1.44
        valid_items = n - i >= BLOCK_SIZE ? BLOCK_SIZE : n - i;
        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state1[i]), c1s, valid_items, 128);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state2[i]), c2s, valid_items, 0);
        //LoadT(temp_storage.loadh).Load(&(p[i]), p_vals, valid_items, (T)0.0f);

        new_local_abs_max1 = -FLT_MAX;
        new_local_abs_max2 = -FLT_MAX;

        //  update: 2.48/1.57 -> 2.51/1.60
        # pragma unroll N_PER_TH
        for(unsigned int j = 0; j < N_PER_TH; j++)
        {
            g_val = float(g_vals[j]);
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[lane_id][c1s[j]]*absmax1[i/BLOCK_SIZE];
            s1_vals[j] = (s1_vals[j]*beta1) + (((1.0f-beta1)*g_val));

            s2_vals[j] = smem_quantiles2[lane_id][c2s[j]]*absmax2[i/BLOCK_SIZE];
            s2_vals[j] = (s2_vals[j]*beta2) + (((1.0f-beta2)*g_val*g_val));

            new_local_abs_max1 = fmaxf(new_local_abs_max1, fabsf(s1_vals[j]));
            new_local_abs_max2 = fmaxf(new_local_abs_max2, fabsf(s2_vals[j]));
        }


        //  reduce: 2.51/1.60 -> 2.67/1.69
        new_local_abs_max1 = BlockReduce1(reduce1).Reduce(new_local_abs_max1, hipcub::Max());
        new_local_abs_max2 = BlockReduce2(reduce2).Reduce(new_local_abs_max2, hipcub::Max());

        if(threadIdx.x == 0)
        {
          smem_exchange1[0] = new_local_abs_max1;
          smem_exchange2[0] = new_local_abs_max2;
        }

        __syncthreads();

        if(threadIdx.x == 0)
        {
          absmax1[i/BLOCK_SIZE] = new_local_abs_max1;
          absmax2[i/BLOCK_SIZE] = new_local_abs_max2;
        }
        else
        {
          new_local_abs_max1 = smem_exchange1[0];
          new_local_abs_max2 = smem_exchange2[0];
        }

        __syncwarp();


        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(p[i]), g_vals, valid_items, (T)0.0f);
        //  reduce: 2.67/1.69 -> 2.67/1.70
        # pragma unroll N_PER_TH
        for(unsigned int j = 0; j < N_PER_TH; j++)
        {
            //p_vals[j] = (T)(((float)p_vals[j]) + ((step_size*(__fdividef(s1_vals[j],(sqrtf(s2_vals[j])+(correction2*eps)))))));
            //if(weight_decay > 0.0f)
            //    p_vals[j] = ((float)p_vals[j])*(1.0f-(lr*weight_decay));
            g_vals[j] = (T)(((float)g_vals[j]) + ((step_size*(__fdividef(s1_vals[j],(sqrtf(s2_vals[j])+(correction2*eps)))))));
            if(weight_decay > 0.0f)
                g_vals[j] = ((float)g_vals[j])*(1.0f-(lr*weight_decay));
        }

        //  store: 0.85/1.44 -> 2.48/1.57
        //StoreT(temp_storage.storeh).Store(&(p[i]), p_vals, valid_items);
        __syncthreads();
        StoreT(temp_storage.storeh).Store(&(p[i]), g_vals, valid_items);
        //  quantizaztion: 2.67/1.70  -> 4.8/4.0
        //  quantizaztion: 2.67/1.70  -> 3.5/3.5
        //  quantizaztion: 2.67/1.70  -> 3.4/3.3
        # pragma unroll N_PER_TH 
        for(unsigned int j = 0; j < N_PER_TH; j++)
        {
            c1s[j] = quantize_2D(smem_quantiles1, lane_id,__fdividef(s1_vals[j],new_local_abs_max1));
            c2s[j] = quantize_2D(smem_quantiles2, lane_id,__fdividef(s2_vals[j],new_local_abs_max2));

            // make sure state1 term has still the same sign after quantization
            // (not needed for state2 term which has only positive values)
            if(signbit(smem_quantiles1[lane_id][c1s[j]]) != signbit(s1_vals[j]))
            {
              if(s1_vals[j] > 0.0f)
                  c1s[j] += 1;
              else
                  c1s[j] -= 1;
            }
        }

        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state1[i]), c1s, valid_items);
        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state2[i]), c2s, valid_items);
    }
}

//==============================================================
//                   TEMPLATE DEFINITIONS
//==============================================================


template __global__ void kEstimateQuantiles(float *__restrict__ const A, float *code, const float offset, const float max_val, const int n);
template __global__ void kEstimateQuantiles(half *__restrict__ const A, float *code, const float offset, const half max_val, const int n);

#define MAKE_PreconditionOptimizer32bit1State(oname, gtype) \
template __global__ void kPreconditionOptimizer32bit1State<gtype, oname, 4096, 4>(gtype* g, gtype* p, \
                float* state1, float *unorm, \
                const float beta1, const float eps, const float weight_decay, \
                const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n); \

MAKE_PreconditionOptimizer32bit1State(MOMENTUM, half)
MAKE_PreconditionOptimizer32bit1State(MOMENTUM, float)
MAKE_PreconditionOptimizer32bit1State(RMSPROP, half)
MAKE_PreconditionOptimizer32bit1State(RMSPROP, float)

#define MAKE_Optimizer32bit1State(oname, gtype) \
template __global__ void kOptimizer32bit1State<gtype, oname>(gtype* g, gtype* p, float* state1, float *unorm, const float max_unorm, const float param_norm, \
    const float beta1, const float eps, const float weight_decay,const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n); \

MAKE_Optimizer32bit1State(MOMENTUM, half)
MAKE_Optimizer32bit1State(MOMENTUM, float)
MAKE_Optimizer32bit1State(RMSPROP, half)
MAKE_Optimizer32bit1State(RMSPROP, float)

#define MAKE_PreconditionOptimizer32bit2State(oname, gtype) \
template __global__ void kPreconditionOptimizer32bit2State<gtype, oname, 4096, 4>(gtype* g, gtype* p,  \
                float* state1, float* state2, float *unorm, \
                const float beta1, const float beta2, const float eps, const float weight_decay, \
                const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n); \

MAKE_PreconditionOptimizer32bit2State(ADAM, half)
MAKE_PreconditionOptimizer32bit2State(ADAM, float)

template __global__ void kOptimizer32bit2State<half, ADAM>(half* g, half* p, float* state1, float* state2, float *unorm, const float max_unorm, const float param_norm,
    const float beta1, const float beta2, const float eps, const float weight_decay,const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n);
template __global__ void kOptimizer32bit2State<float, ADAM>(float* g, float* p, float* state1, float* state2, float *unorm, const float max_unorm, const float param_norm,
    const float beta1, const float beta2, const float eps, const float weight_decay,const int step, const float lr, const bool is_sparse, const float gnorm_scale, const int n);

#define MAKE_PreconditionStatic8bit1State(oname, gtype) \
template __global__ void kPreconditionOptimizerStatic8bit1State<gtype, oname>(gtype* p, gtype* __restrict__ const g, unsigned char*__restrict__  const state1,  \
                float *unorm,  \
                const float beta1,  \
                const float eps, const int step,  \
                float* __restrict__ const quantiles1,  \
                float* max1, float* new_max1,  \
                const float gnorm_scale,  \
                const int n); \

MAKE_PreconditionStatic8bit1State(MOMENTUM, half)
MAKE_PreconditionStatic8bit1State(MOMENTUM, float)
MAKE_PreconditionStatic8bit1State(RMSPROP, half)
MAKE_PreconditionStatic8bit1State(RMSPROP, float)

#define MAKE_optimizerStatic8bit1State(oname, gtype) \
template __global__ void kOptimizerStatic8bit1State<gtype, oname>(gtype* p, gtype* const g, unsigned char* state1,  \
                const float *unorm, const float max_unorm, const float param_norm, \
                const float beta1,  \
                const float eps, const int step, const float lr, \
                float* __restrict__ const quantiles1,  \
                float* max1, float* new_max1,  \
                float weight_decay, \
                const float gnorm_scale,  \
                const int n); \

MAKE_optimizerStatic8bit1State(MOMENTUM, half)
MAKE_optimizerStatic8bit1State(MOMENTUM, float)
MAKE_optimizerStatic8bit1State(RMSPROP, half)
MAKE_optimizerStatic8bit1State(RMSPROP, float)

#define MAKE_PreconditionStatic8bit2State(oname, gtype) \
template __global__ void kPreconditionOptimizerStatic8bit2State<gtype, oname>(gtype* p, gtype* __restrict__ const g, unsigned char*__restrict__  const state1, unsigned char* __restrict__ const state2, \
                float *unorm, \
                const float beta1, const float beta2, \
                const float eps, const int step,  \
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                const float gnorm_scale,  \
                const int n); \

MAKE_PreconditionStatic8bit2State(ADAM, half)
MAKE_PreconditionStatic8bit2State(ADAM, float)

#define MAKE_optimizerStatic8bit2State(oname, gtype) \
template __global__ void kOptimizerStatic8bit2State<gtype, oname>(gtype* p, gtype* const g, unsigned char* state1, unsigned char* state2, \
                const float *unorm, const float max_unorm, const float param_norm, \
                const float beta1, const float beta2, \
                const float eps, const int step, const float lr, \
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                float weight_decay, \
                const float gnorm_scale,  \
                const int n); \

MAKE_optimizerStatic8bit2State(ADAM, half)
MAKE_optimizerStatic8bit2State(ADAM, float)

template __global__ void kPercentileClipping<float, 2048, 4>(float * __restrict__ g, float *gnorm_vec, int step, const int n);
template __global__ void kPercentileClipping<half, 2048, 4>(half * __restrict__ g, float *gnorm_vec, int step, const int n);

template __global__ void kQuantizeBlockwise<half, 4096, 4>(float * code, half * __restrict__ const A, float *absmax, unsigned char *out, const int n);
template __global__ void kQuantizeBlockwise<float, 4096, 4>(float * code, float * __restrict__ const A, float *absmax, unsigned char *out, const int n);

template __global__ void kDequantizeBlockwise<half, 4096, 1024, 4>(float *code, unsigned char * __restrict__ const A, float * __restrict__ const absmax, half *out, const int n);
template __global__ void kDequantizeBlockwise<float, 4096, 1024, 4>(float *code, unsigned char * __restrict__ const A, float * __restrict__ const absmax, float *out, const int n);
template __global__ void kDequantizeBlockwise<half, 2048, 512, 4>(float *code, unsigned char * __restrict__ const A, float * __restrict__ const absmax, half *out, const int n);
template __global__ void kDequantizeBlockwise<float, 2048, 512, 4>(float *code, unsigned char * __restrict__ const A, float * __restrict__ const absmax, float *out, const int n);



#define MAKE_OptimizerStatic8bit2StateBlockwise(oname, gtype, block_size, num_per_thread) \
template __global__ void kOptimizerStatic8bit2StateBlockwise<gtype, oname, block_size, num_per_thread>(gtype* p, gtype* __restrict__ const g, unsigned char* state1, unsigned char* state2, \
                const float beta1, const float beta2, \
                const float eps, const int step, const float lr, \
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2, \
                float* absmax1, float* absmax2,  \
                float weight_decay, \
                const float gnorm_scale, const int n); \


MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, float, 2048, 8)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, half, 2048, 8)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, float, 2048, 4)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, half, 2048, 4)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, float, 4096, 4)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, half, 4096, 4)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, float, 4096, 8)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, half, 4096, 8)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, float, 4096, 16)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, half, 4096, 16)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, float, 8192, 8)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, half, 8192, 8)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, float, 8192, 16)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, half, 8192, 16)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, float, 4096, 32)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, half, 4096, 32)
